#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_2320
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2320_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2320(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2320_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2320_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_375
// Description:	Constant
// Input:
// Output:
//	- name: Constant_375_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_375(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_375_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_375_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2992
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2992_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2992(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2992_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2992_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2407
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2407_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2407(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2407_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2407_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_14
// Description:	Constant
// Input:
// Output:
//	- name: Constant_14_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_14(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_14_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_14_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2413
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2413_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2413(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2413_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2413_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2774
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2774_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2774(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2774_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2774_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2314
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2314_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2314(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2314_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2314_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2964
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2964_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2964(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2964_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2964_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_416
// Description:	Constant
// Input:
// Output:
//	- name: Constant_416_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_416(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_416_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_416_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[2304];
  bin_file.read(tmp_mem, 2304);
  hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2906
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2906_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2906(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2906_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2906_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_467_0	type: float	shape: Shape{1,
// 96, 32, 32}
//	- name: Constant_2014_0	type: float	shape: Shape{32, 96, 1, 1}
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_434_0	type: float	shape: Shape{5, 5, 96, 1}
//	- name: Constant_58_0	type: float	shape: Shape{3, 3, 96, 1}
//	- name: Constant_2029_0	type: float	shape: Shape{32, 96, 1, 1}
//	- name: Constant_2026_0	type: float	shape: Shape{32, 96, 1, 1}
//	- name: Constant_355_0	type: float	shape: Shape{3, 3, 96, 1}
//	- name: Constant_2023_0	type: float	shape: Shape{32, 96, 1, 1}
// Output:
//	- name: Convolution_471_0	type: float	shape: Shape{1, 32, 32,
// 32}
//	- name: DepthwiseConv2dNative_480_0	type: float	shape: Shape{1,
// 96, 32, 32}
//	- name: DepthwiseConv2dNative_479_0	type: float	shape: Shape{1,
// 96, 32, 32}
//	- name: Convolution_476_0	type: float	shape: Shape{1, 32, 32,
// 32}
//	- name: Convolution_474_0	type: float	shape: Shape{1, 32, 32,
// 32}
//	- name: DepthwiseConv2dNative_472_0	type: float	shape: Shape{1,
// 96, 32, 32}
//	- name: Convolution_478_0	type: float	shape: Shape{1, 32, 32,
// 32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_471<<<dim3(1, 32, 2), dim3(16,
// 1, 8), 0, 0>>>(BatchNormInference_467_0, Constant_2014_0, Convolution_471_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_480<<<dim3(768,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_468_0, Constant_434_0,
// DepthwiseConv2dNative_480_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_479<<<dim3(768,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_468_0, Constant_58_0,
// DepthwiseConv2dNative_479_0);
// Convolution_float_float_float_cuda_Convolution_476<<<dim3(1, 32, 2), dim3(16,
// 1, 8), 0, 0>>>(Relu_468_0, Constant_2029_0, Convolution_476_0);
// Convolution_float_float_float_cuda_Convolution_474<<<dim3(1, 32, 2), dim3(16,
// 1, 8), 0, 0>>>(Relu_468_0, Constant_2026_0, Convolution_474_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472<<<dim3(768,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_468_0, Constant_355_0,
// DepthwiseConv2dNative_472_0);
// Convolution_float_float_float_cuda_Convolution_478<<<dim3(1, 32, 2), dim3(16,
// 1, 8), 0, 0>>>(Relu_468_0, Constant_2023_0, Convolution_478_0); Deduped
// function map: <src_function_name : deduped_function_name>
// Convolution_float_float_float_cuda_Convolution_476 :
// Convolution_float_float_float_cuda_Convolution_471
// Convolution_float_float_float_cuda_Convolution_474 :
// Convolution_float_float_float_cuda_Convolution_471
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_472 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_479
// Convolution_float_float_float_cuda_Convolution_478 :
// Convolution_float_float_float_cuda_Convolution_471

// Node name:	Convolution_471
// Description:	Convolution
// Input:
//	- name: BatchNormInference_467_0	type: float	shape: Shape{1,
// 96, 32, 32}
//	- name: Constant_2014_0	type: float	shape: Shape{32, 96, 1, 1}
// Output:
//	- name: Convolution_471_0	type: float	shape: Shape{1, 32, 32,
// 32}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_471_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(16, 1, 8);
  const dim3 gridDim(1, 32, 2);
  const dim3 threadIdx(thread_id % 16, 0, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 32, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute_local[4];

#pragma unroll
      for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
        compute_local[xx_c_init] = 0.000000e+00f;
        compute_local[(xx_c_init + 2)] = 0.000000e+00f;
      }
#pragma unroll
      for (int rc_outer = 0; rc_outer < 6; ++rc_outer) {
        __syncthreads();
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              ((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 4)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                    ((((((int)threadIdx.x) * 4) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      5) *
                     1024)) +
                   (((int)blockIdx.y) * 32)) +
                  (((((int)threadIdx.x) * 4) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   31))];
        }
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
          input1_shared[(
              ((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input1[(
                  ((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >>
                      4) *
                     96)) +
                   (rc_outer * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) &
                   15))];
        }
        __syncthreads();
#pragma unroll
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
#pragma unroll
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute_local[xx_c] =
                (compute_local[xx_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared[((((int)threadIdx.z) * 16) + rc_inner)]));
            compute_local[(xx_c + 2)] =
                (compute_local[(xx_c + 2)] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared[(((((int)threadIdx.z) * 16) + rc_inner) +
                                 128)]));
          }
        }
      }
#pragma unroll
      for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2;
           ++xx_inner_inner_inner) {
        compute[(((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
                   (((int)blockIdx.y) * 32)) +
                  (((int)threadIdx.x) * 2)) +
                 xx_inner_inner_inner)] = compute_local[xx_inner_inner_inner];
        compute[(
            (((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
               (((int)blockIdx.y) * 32)) +
              (((int)threadIdx.x) * 2)) +
             xx_inner_inner_inner) +
            8192)] = compute_local[(xx_inner_inner_inner + 2)];
      }
    }
  }
}
// Node name:	DepthwiseConv2dNative_480
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_434_0	type: float	shape: Shape{5, 5, 96, 1}
// Output:
//	- name: DepthwiseConv2dNative_480_0	type: float	shape: Shape{1,
// 96, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_480_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(768, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 96;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 96;
  const int num_outputs = 98304;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_479
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: Constant_58_0	type: float	shape: Shape{3, 3, 96, 1}
// Output:
//	- name: DepthwiseConv2dNative_479_0	type: float	shape: Shape{1,
// 96, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_479_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(768, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 96;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 96;
  const int num_outputs = 98304;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
__device__ __forceinline__ void
Fuse_2_DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_479_block_kernel(
    float *input0, float *input1, float *input2, float *output0, float *output1,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(768, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  // float *input = input0;
  // float *filter = input1;
  // float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 96;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 96;
  const int num_outputs = 98304;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum0 = static_cast<S>(0);
    S sum1 = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum0 += static_cast<S>(__ldg(input0 + input_offset)) *
                  static_cast<S>(__ldg(input1 + filter_offset));
          sum1 += static_cast<S>(__ldg(input0 + input_offset)) *
                  static_cast<S>(__ldg(input2 + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum0 += static_cast<S>(__ldg(input0 + input_offset)) *
                    static_cast<S>(__ldg(input1 + filter_offset));
            sum1 += static_cast<S>(__ldg(input0 + input_offset)) *
                    static_cast<S>(__ldg(input2 + filter_offset));
          }
        }
      }
    }

    output0[thread_id] = static_cast<S>(sum0);
    output1[thread_id] = static_cast<S>(sum1);
  }
}
__device__ __forceinline__ void
fused3_Convolution_float_float_float_cuda_Convolution_471_block_kernel1(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1, float *output2, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(16, 1, 8);
  const dim3 gridDim(1, 32, 2);
  const dim3 threadIdx(thread_id % 16, 0, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 32, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  float *input1_shared1 = (float *)(shared_buffer + 3072);
  float *input1_shared2 = (float *)(shared_buffer + 4096);
  {
    float *compute = output0;
    {
      float compute_local[4];
      float compute_local1[4];
      float compute_local2[4];

#pragma unroll
      for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
        compute_local[xx_c_init] = 0.000000e+00f;
        compute_local[(xx_c_init + 2)] = 0.000000e+00f;
        compute_local1[xx_c_init] = 0.000000e+00f;
        compute_local1[(xx_c_init + 2)] = 0.000000e+00f;
        compute_local2[xx_c_init] = 0.000000e+00f;
        compute_local2[(xx_c_init + 2)] = 0.000000e+00f;
      }
#pragma unroll
      for (int rc_outer = 0; rc_outer < 6; ++rc_outer) {
        __syncthreads();
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              ((((int)threadIdx.z) * 64) + (((int)threadIdx.x) * 4)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                    ((((((int)threadIdx.x) * 4) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      5) *
                     1024)) +
                   (((int)blockIdx.y) * 32)) +
                  (((((int)threadIdx.x) * 4) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   31))];
        }
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
          input1_shared[(
              ((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input1[(
                  ((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >>
                      4) *
                     96)) +
                   (rc_outer * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) &
                   15))];
          input1_shared1[(
              ((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input2[(
                  ((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >>
                      4) *
                     96)) +
                   (rc_outer * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) &
                   15))];
          input1_shared2[(
              ((((int)threadIdx.z) * 32) + (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input3[(
                  ((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) >>
                      4) *
                     96)) +
                   (rc_outer * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) &
                   15))];
        }
        __syncthreads();
#pragma unroll
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
#pragma unroll
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute_local[xx_c] =
                (compute_local[xx_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared[((((int)threadIdx.z) * 16) + rc_inner)]));
            compute_local[(xx_c + 2)] =
                (compute_local[(xx_c + 2)] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared[(((((int)threadIdx.z) * 16) + rc_inner) +
                                 128)]));
            compute_local1[xx_c] =
                (compute_local1[xx_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared1[((((int)threadIdx.z) * 16) + rc_inner)]));
            compute_local1[(xx_c + 2)] =
                (compute_local1[(xx_c + 2)] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared1[(((((int)threadIdx.z) * 16) + rc_inner) +
                                  128)]));
            compute_local2[xx_c] =
                (compute_local2[xx_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared2[((((int)threadIdx.z) * 16) + rc_inner)]));
            compute_local2[(xx_c + 2)] =
                (compute_local2[(xx_c + 2)] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.x) * 2)) + xx_c)] *
                  input1_shared2[(((((int)threadIdx.z) * 16) + rc_inner) +
                                  128)]));
          }
        }
      }
#pragma unroll
      for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2;
           ++xx_inner_inner_inner) {
        compute[(((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
                   (((int)blockIdx.y) * 32)) +
                  (((int)threadIdx.x) * 2)) +
                 xx_inner_inner_inner)] = compute_local[xx_inner_inner_inner];
        compute[(
            (((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
               (((int)blockIdx.y) * 32)) +
              (((int)threadIdx.x) * 2)) +
             xx_inner_inner_inner) +
            8192)] = compute_local[(xx_inner_inner_inner + 2)];
        output1[(((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
                   (((int)blockIdx.y) * 32)) +
                  (((int)threadIdx.x) * 2)) +
                 xx_inner_inner_inner)] = compute_local1[xx_inner_inner_inner];
        output1[(
            (((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
               (((int)blockIdx.y) * 32)) +
              (((int)threadIdx.x) * 2)) +
             xx_inner_inner_inner) +
            8192)] = compute_local1[(xx_inner_inner_inner + 2)];
        output2[(((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
                   (((int)blockIdx.y) * 32)) +
                  (((int)threadIdx.x) * 2)) +
                 xx_inner_inner_inner)] = compute_local2[xx_inner_inner_inner];
        output2[(
            (((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) +
               (((int)blockIdx.y) * 32)) +
              (((int)threadIdx.x) * 2)) +
             xx_inner_inner_inner) +
            8192)] = compute_local2[(xx_inner_inner_inner + 2)];
      }
    }
  }
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_DepthwiseConv2dNative_DepthwiseConv2dNative_Convolution_Convolution_DepthwiseConv2dNative_Convolution_0(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *output0,
    float *output1, float *output2, float *output3, float *output4,
    float *output5, float *output6) {

  __shared__ char shared_buffer[5120];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_471_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    fused3_Convolution_float_float_float_cuda_Convolution_471_block_kernel1(
        input2, input5, input6, input8, output3, output4, output6, threadIdx.x,
        blockIdx.x - 64, shared_buffer);
  }
  // else if((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
  //{
  // Convolution_float_float_float_cuda_Convolution_471_block_kernel(input2,
  // input6, output4, threadIdx.x, blockIdx.x - 128, shared_buffer);
  //}
  // else if((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255)
  //{
  // Convolution_float_float_float_cuda_Convolution_471_block_kernel(input2,
  // input8, output6, threadIdx.x, blockIdx.x - 192, shared_buffer);
  //}
  else if ((int)blockIdx.x >= 256 - 128 && (int)blockIdx.x <= 1023 - 128) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_480_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 256 + 128,
        shared_buffer);
  } else if ((int)blockIdx.x >= 1024 - 128 &&
             (int)blockIdx.x <= 1024 + 767 - 128) {
    Fuse_2_DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_479_block_kernel(
        input2, input4, input7, output2, output5, threadIdx.x,
        blockIdx.x - 1024 + 128, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_DepthwiseConv2dNative_DepthwiseConv2dNative_Convolution_Convolution_DepthwiseConv2dNative_Convolution_0_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *output0,
    float *output1, float *output2, float *output3, float *output4,
    float *output5, float *output6) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Convolution_DepthwiseConv2dNative_DepthwiseConv2dNative_Convolution_Convolution_DepthwiseConv2dNative_Convolution_0<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      output0, output1, output2, output3, output4, output5, output6);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1649_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2680_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1650_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2683_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1655_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Convolution_1657_0	type: float	shape: Shape{1, 128, 8,
// 8}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1655<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1649_0, Constant_2680_0,
// Convolution_1655_0);
// Convolution_float_float_float_cuda_Convolution_1657<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1650_0, Constant_2683_0,
// Convolution_1657_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_1657 :
// Convolution_float_float_float_cuda_Convolution_1655

// Node name:	Convolution_1655
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1649_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2680_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1655_0	type: float	shape: Shape{1, 128, 8,
// 8}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_1655_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute_local[1];

      compute_local[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] = compute_local[0];
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_164(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_1655_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_1655_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_164_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_164<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2766_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1414_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Convolution_1416_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Constant_2932_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1417_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Relu_1420_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1418_0	type: float	shape: Shape{1,
// 128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Relu_53<<<dim3(16, 1, 1),
// dim3(512, 1, 1), 0, 0>>>(Convolution_1414_0, Constant_2766_0, Relu_1420_0,
// BatchNormInference_1417_0); Add_float_float_float_cuda_Add_2547<<<dim3(16, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1416_0, Constant_2932_0,
// BatchNormInference_1418_0); Deduped function map: <src_function_name :
// deduped_function_name>

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1414_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Constant_2766_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1420_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1417_0	type: float	shape: Shape{1,
// 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2544<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1414_0, Constant_2766_0, BatchNormInference_1417_0);
// Relu_float_float_cuda_Relu_1420<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1417_0, Relu_1420_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_cuda_Add_Relu_53_block_kernel(
    float *input0, float *input1, float *output0, float *output1, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = relu(temp0);
  output1[tid] = temp0;
  output0[tid] = temp1;
}
// Node name:	Add_2547
// Description:	Add
// Input:
//	- name: Convolution_1416_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Constant_2932_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1418_0	type: float	shape: Shape{1,
// 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_2547_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_131(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1, float *output2) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    FusedKernel_float_float_float_float_cuda_Add_Relu_53_block_kernel(
        input1, input0, output1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_2547_block_kernel(
        input2, input3, output2, threadIdx.x, blockIdx.x - 16, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_131_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1, float *output2) {
  BlockFusionKernel_float_float_float_float_float_float_float_cuda_fused_kernel_Add_131<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1, output2);
}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_1666_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Constant_2963_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2964_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1668_0	type: float	shape: Shape{1, 128, 8,
// 8}
// Output:
//	- name: BatchNormInference_1669_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: BatchNormInference_1670_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Relu_1672_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2688<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1666_0, Constant_2963_0, BatchNormInference_1669_0);
// FusedKernel_float_float_float_float_cuda_Add_Relu_69<<<dim3(16, 1, 1),
// dim3(512, 1, 1), 0, 0>>>(Convolution_1668_0, Constant_2964_0, Relu_1672_0,
// BatchNormInference_1670_0); Deduped function map: <src_function_name :
// deduped_function_name>

// Node name:	Add_2688
// Description:	Add
// Input:
//	- name: Convolution_1666_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Constant_2963_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1669_0	type: float	shape: Shape{1,
// 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_2688_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1668_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Constant_2964_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1672_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1670_0	type: float	shape: Shape{1,
// 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2691<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1668_0, Constant_2964_0, BatchNormInference_1670_0);
// Relu_float_float_cuda_Relu_1672<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1670_0, Relu_1672_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_cuda_Add_Relu_69_block_kernel(
    float *input0, float *input1, float *output0, float *output1, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = relu(temp0);
  output1[tid] = temp0;
  output0[tid] = temp1;
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_cuda_Add_fused_kernel_167(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1, float *output2) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Add_float_float_float_cuda_Add_2688_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    FusedKernel_float_float_float_float_cuda_Add_Relu_69_block_kernel(
        input3, input2, output2, output1, threadIdx.x, blockIdx.x - 16, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_cuda_Add_fused_kernel_167_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1, float *output2) {
  BlockFusionKernel_float_float_float_float_float_float_float_cuda_Add_fused_kernel_167<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1, output2);
}
