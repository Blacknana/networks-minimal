#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	 BlockFusion
// Input:
//	- name: AvgPool_1609_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1613_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2662_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3164_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1611_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2656_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3160_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1612_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2659_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3162_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1610_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1614_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1635_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1633_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1634_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1616_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_1614<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(AvgPool_1609_0, AvgPool_1609_0, Add_1614_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1613_0, Constant_2662_0,
// Constant_3164_0, Relu_1635_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3159<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1611_0, Constant_2656_0,
// Constant_3160_0, Relu_1633_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3161<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1612_0, Constant_2659_0,
// Constant_3162_0, Relu_1634_0); Relu_float_float_cuda_Relu_1616<<<dim3(16, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Slice_1610_0, Relu_1616_0); Deduped function
// map: <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3159 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3161 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163

// Node name:	Add_1614
// Description:	Add
// Input:
//	- name: AvgPool_1609_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1609_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1614_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1614_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Matched_Pattern_3163
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1613_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2662_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3164_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1635_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[1];

      compute1[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
// Node name:	Relu_1616
// Description:	Relu
// Input:
//	- name: Slice_1610_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1616_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_1616_block_kernel(float *input0, float *output0,
                                             int thread_id, int block_id,
                                             char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_Relu_160(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Add_float_float_float_cuda_Add_1614_block_kernel(
        input0, input0, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Relu_float_float_cuda_Relu_1616_block_kernel(
        input10, output4, threadIdx.x, blockIdx.x - 16, shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163_block_kernel(
        input1, input2, input3, output1, threadIdx.x, blockIdx.x - 32,
        shared_buffer);
  } else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163_block_kernel(
        input4, input5, input6, output2, threadIdx.x, blockIdx.x - 96,
        shared_buffer);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3163_block_kernel(
        input7, input8, input9, output3, threadIdx.x, blockIdx.x - 160,
        shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_Relu_160_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_Relu_160<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	Constant_220
// Description:	Constant
// Input:
// Output:
//	- name: Constant_220_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_220(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_220_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_220_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3200];
  bin_file.read(tmp_mem, 3200);
  hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2632
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2632_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2632(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2632_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2632_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_348
// Description:	Constant
// Input:
// Output:
//	- name: Constant_348_0	type: float	shape: Shape{7, 7, 64, 1}
void Constant_float_cuda_Constant_348(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_348_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_348_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12544];
  bin_file.read(tmp_mem, 12544);
  hipMemcpyAsync(output0, tmp_mem, 12544, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2587
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2587_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2587(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2587_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2587_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2347
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2347_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2347(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2347_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2347_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_313
// Description:	Constant
// Input:
// Output:
//	- name: Constant_313_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_313(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_313_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_313_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2203
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2203_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2203(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2203_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2203_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2437
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2437_0	type: float	shape: Shape{64, 384, 1, 1}
void Constant_float_cuda_Constant_2437(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2437_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2437_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[98304];
  bin_file.read(tmp_mem, 98304);
  hipMemcpyAsync(output0, tmp_mem, 98304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2263
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2263_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2263(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2263_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2263_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2605
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2605_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2605(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2605_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2605_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_778_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2203_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3032_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_779_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2206_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3034_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_801_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2209_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_803_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2215_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_802_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2212_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Relu_799_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_800_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_807_0	type: float	shape: Shape{1, 32, 32,
// 32}
//	- name: Convolution_811_0	type: float	shape: Shape{1, 32, 32,
// 32}
//	- name: Convolution_809_0	type: float	shape: Shape{1, 32, 32,
// 32}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3031<<<dim3(2,
// 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_778_0, Constant_2203_0,
// Constant_3032_0, Relu_799_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3033<<<dim3(2,
// 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_779_0, Constant_2206_0,
// Constant_3034_0, Relu_800_0);
// Convolution_float_float_float_cuda_Convolution_807<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_801_0, Constant_2209_0,
// Convolution_807_0);
// Convolution_float_float_float_cuda_Convolution_811<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_803_0, Constant_2215_0,
// Convolution_811_0);
// Convolution_float_float_float_cuda_Convolution_809<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_802_0, Constant_2212_0,
// Convolution_809_0); Deduped function map: <src_function_name :
// deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3033 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3031
// Convolution_float_float_float_cuda_Convolution_811 :
// Convolution_float_float_float_cuda_Convolution_807
// Convolution_float_float_float_cuda_Convolution_809 :
// Convolution_float_float_float_cuda_Convolution_807

// Node name:	Matched_Pattern_3031
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_778_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2203_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3032_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_799_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3031_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 2, 8);
  const dim3 gridDim(2, 16, 2);
  const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute1[2];

      for (int ff_init = 0; ff_init < 2; ++ff_init) {
        compute1[ff_init] = 0.000000e+00f;
      }
      for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) +
               (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                      (((int)threadIdx.y) * 1024)) +
                     (((int)blockIdx.y) * 64)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      4) *
                     32)) +
                   (((int)blockIdx.x) * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   15))];
        }
        input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       ((int)threadIdx.x))] =
            input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                      (((int)threadIdx.y) * 32)) +
                     (rc_outer * 16)) +
                    ((int)threadIdx.x))];
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff = 0; ff < 2; ++ff) {
            compute1[ff] =
                (compute1[ff] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.y) * 16)) +
                      ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 32) + (ff * 16)) +
                                 rc_inner)]));
          }
        }
      }
      for (int i1_inner_inner_inner = 0; i1_inner_inner_inner < 2;
           ++i1_inner_inner_inner) {
        compute[(
            ((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) +
                (i1_inner_inner_inner * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (((int)threadIdx.y) * 32)) +
             (((int)blockIdx.x) * 16)) +
            ((int)threadIdx.x))] =
            max((compute1[i1_inner_inner_inner] +
                 input2[(((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) +
                         i1_inner_inner_inner)]),
                0.000000e+00f);
      }
    }
  }
}
// Node name:	Convolution_807
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_801_0	type: float	shape: Shape{1,
// 32, 32, 32}
//	- name: Constant_2209_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_807_0	type: float	shape: Shape{1, 32, 32,
// 32}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_807_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 2, 8);
  const dim3 gridDim(2, 16, 2);
  const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute_local[2];

      for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
        compute_local[ff_c_init] = 0.000000e+00f;
      }
      for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) +
               (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                      (((int)threadIdx.y) * 1024)) +
                     (((int)blockIdx.y) * 64)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      4) *
                     32)) +
                   (((int)blockIdx.x) * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   15))];
        }
        input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       ((int)threadIdx.x))] =
            input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                      (((int)threadIdx.y) * 32)) +
                     (rc_outer * 16)) +
                    ((int)threadIdx.x))];
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[ff_c] =
                (compute_local[ff_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.y) * 16)) +
                      ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) +
                                 rc_inner)]));
          }
        }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2;
           ++ff_inner_inner_inner) {
        compute[(
            ((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) +
                (ff_inner_inner_inner * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (((int)threadIdx.y) * 32)) +
             (((int)blockIdx.x) * 16)) +
            ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
      }
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_41(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *input11, float *output0, float *output1,
    float *output2, float *output3, float *output4) {

  __shared__ char shared_buffer[3072];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3031_block_kernel(
        input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3031_block_kernel(
        input3, input4, input5, output1, threadIdx.x, blockIdx.x - 64,
        shared_buffer);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191) {
    Convolution_float_float_float_cuda_Convolution_807_block_kernel(
        input6, input7, output2, threadIdx.x, blockIdx.x - 128, shared_buffer);
  } else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255) {
    Convolution_float_float_float_cuda_Convolution_807_block_kernel(
        input8, input9, output3, threadIdx.x, blockIdx.x - 192, shared_buffer);
  } else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319) {
    Convolution_float_float_float_cuda_Convolution_807_block_kernel(
        input10, input11, output4, threadIdx.x, blockIdx.x - 256,
        shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_41_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *input11, float *output0, float *output1,
    float *output2, float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Convolution_Convolution_Convolution_41<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, input11, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_885_0	type: float	shape: Shape{1, 192, 32, 32}
//	- name: Constant_2260_0	type: float	shape: Shape{64, 192, 1, 1}
//	- name: Constant_889_0	type: float	shape: Shape{}
// Output:
//	- name: Convolution_887_0	type: float	shape: Shape{1, 64, 32,
// 32}
//	- name: Pad_890_0	type: float	shape: Shape{1, 192, 33, 33}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_887<<<dim3(1, 32, 2), dim3(16,
// 1, 16), 0, 0>>>(Relu_885_0, Constant_2260_0, Convolution_887_0);
// Pad_float_float_float_cuda_Pad_890<<<dim3(3267, 1, 1), dim3(64, 1, 1), 0,
// 0>>>(Relu_885_0, Constant_889_0, Pad_890_0); Deduped function map:
// <src_function_name : deduped_function_name>

// Node name:	Convolution_887
// Description:	Convolution
// Input:
//	- name: Relu_885_0	type: float	shape: Shape{1, 192, 32, 32}
//	- name: Constant_2260_0	type: float	shape: Shape{64, 192, 1, 1}
// Output:
//	- name: Convolution_887_0	type: float	shape: Shape{1, 64, 32,
// 32}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_887_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 1, 16);
  const dim3 gridDim(1, 32, 2);
  const dim3 threadIdx(thread_id % 16, 0, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 32, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 3072);
  {
    float *compute = output0;
    {
      float compute_local[4];

#pragma unroll
      for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
        compute_local[ff_c_init] = 0.000000e+00f;
        compute_local[(ff_c_init + 2)] = 0.000000e+00f;
      }
#pragma unroll
      for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
        __syncthreads();
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              ((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  (((rc_outer * 24576) +
                    (((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      5) *
                     1024)) +
                   (((int)blockIdx.y) * 32)) +
                  ((((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   31))];
        }
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
          input1_shared[(
              ((((int)threadIdx.z) * 48) + (((int)threadIdx.x) * 3)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input1[(
                  ((((((int)blockIdx.z) * 6144) + (((int)threadIdx.z) * 384)) +
                    ((((((int)threadIdx.x) * 3) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) /
                      24) *
                     192)) +
                   (rc_outer * 24)) +
                  (((((int)threadIdx.x) * 3) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) %
                   24))];
        }
        __syncthreads();
#pragma unroll
        for (int rc_inner = 0; rc_inner < 24; ++rc_inner) {
#pragma unroll
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[ff_c] =
                (compute_local[ff_c] +
                 (pad_temp_shared[((rc_inner * 32) + ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 48) + (ff_c * 24)) +
                                 rc_inner)]));
            compute_local[(ff_c + 2)] =
                (compute_local[(ff_c + 2)] +
                 (pad_temp_shared[(((rc_inner * 32) + ((int)threadIdx.x)) +
                                   16)] *
                  input1_shared[(((((int)threadIdx.z) * 48) + (ff_c * 24)) +
                                 rc_inner)]));
          }
        }
      }
#pragma unroll
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2;
           ++ff_inner_inner_inner) {
        compute[(((((((int)blockIdx.z) * 32768) + (((int)threadIdx.z) * 2048)) +
                   (ff_inner_inner_inner * 1024)) +
                  (((int)blockIdx.y) * 32)) +
                 ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
        compute[(
            (((((((int)blockIdx.z) * 32768) + (((int)threadIdx.z) * 2048)) +
               (ff_inner_inner_inner * 1024)) +
              (((int)blockIdx.y) * 32)) +
             ((int)threadIdx.x)) +
            16)] = compute_local[(ff_inner_inner_inner + 2)];
      }
    }
  }
}
// Node name:	Pad_890
// Description:	Pad
// Input:
//	- name: Relu_885_0	type: float	shape: Shape{1, 192, 32, 32}
//	- name: Constant_889_0	type: float	shape: Shape{}
// Output:
//	- name: Pad_890_0	type: float	shape: Shape{1, 192, 33, 33}
__device__ __forceinline__ void Pad_float_float_float_cuda_Pad_890_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 64) {
    return;
  }
  const dim3 blockDim(64, 1, 1);
  const dim3 gridDim(3267, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  float *in = input0;
  float *pad = input1;
  float *out = output0;
  if (tid < 209088) {
    size_t input_shape0 = 1;
    size_t input_shape1 = 192;
    size_t input_shape2 = 32;
    size_t input_shape3 = 32;
    uint32_t input_strides0 = 196608;
    uint32_t input_strides1 = 1024;
    uint32_t input_strides2 = 32;
    uint32_t input_strides3 = 1;
    uint32_t output_strides0 = 209088;
    uint32_t output_strides1 = 1089;
    uint32_t output_strides2 = 33;
    uint32_t output_strides3 = 1;
    uint32_t padding_below0 = 0;
    uint32_t padding_below1 = 0;
    uint32_t padding_below2 = 0;
    uint32_t padding_below3 = 0;
    uint32_t padding_interior0 = 0;
    uint32_t padding_interior1 = 0;
    uint32_t padding_interior2 = 0;
    uint32_t padding_interior3 = 0;
    bool in_bounds = true;
    uint32_t output_pixel = tid;
    uint32_t input_pixel = 0;
    int32_t input, input_dil;
    input_dil = output_pixel / output_strides0 - padding_below0;
    input = input_dil / (padding_interior0 + 1);
    input_dil %= (padding_interior0 + 1);
    in_bounds =
        in_bounds && (input >= 0) && (input < input_shape0) && (input_dil == 0);
    input_pixel += input * input_strides0;
    output_pixel %= output_strides0;
    input_dil = output_pixel / output_strides1 - padding_below1;
    input = input_dil / (padding_interior1 + 1);
    input_dil %= (padding_interior1 + 1);
    in_bounds =
        in_bounds && (input >= 0) && (input < input_shape1) && (input_dil == 0);
    input_pixel += input * input_strides1;
    output_pixel %= output_strides1;
    input_dil = output_pixel / output_strides2 - padding_below2;
    input = input_dil / (padding_interior2 + 1);
    input_dil %= (padding_interior2 + 1);
    in_bounds =
        in_bounds && (input >= 0) && (input < input_shape2) && (input_dil == 0);
    input_pixel += input * input_strides2;
    output_pixel %= output_strides2;
    input_dil = output_pixel / output_strides3 - padding_below3;
    input = input_dil / (padding_interior3 + 1);
    input_dil %= (padding_interior3 + 1);
    in_bounds =
        in_bounds && (input >= 0) && (input < input_shape3) && (input_dil == 0);
    input_pixel += input * input_strides3;
    out[tid] = (in_bounds) ? in[input_pixel] : *pad;
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Pad_54(
    float *input0, float *input1, float *input2, float *output0,
    float *output1) {

  __shared__ char shared_buffer[6144];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_887_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 3330) {
    Pad_float_float_float_cuda_Pad_890_block_kernel(
        input0, input2, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Pad_54_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_cuda_Convolution_Pad_54<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_1343_0	type: float	shape: Shape{1, 64, 8,
// 8}
//	- name: Convolution_1349_0	type: float	shape: Shape{1, 64, 8,
// 8}
//	- name: Relu_1350_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_218_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: MaxPool_1351_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1325_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: AvgPool_1352_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1326_0	type: float	shape: Shape{1,
// 128, 8, 8}
// Output:
//	- name: Concat_1353_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1354_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Add_1355_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1356_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Concat_float_float_float_cuda_Concat_1353<<<dim3(16, 1, 1), dim3(512, 1, 1),
// 0, 0>>>(Convolution_1343_0, Convolution_1349_0, Concat_1353_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1354<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1350_0, Constant_218_0,
// DepthwiseConv2dNative_1354_0); Add_float_float_float_cuda_Add_1355<<<dim3(16,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(MaxPool_1351_0, BatchNormInference_1325_0,
// Add_1355_0); Add_float_float_float_cuda_Add_1356<<<dim3(16, 1, 1), dim3(512,
// 1, 1), 0, 0>>>(AvgPool_1352_0, BatchNormInference_1326_0, Add_1356_0);
// Deduped function map: <src_function_name : deduped_function_name>
// Add_float_float_float_cuda_Add_1356 : Add_float_float_float_cuda_Add_1355

// Node name:	Concat_1353
// Description:	Concat
// Input:
//	- name: Convolution_1343_0	type: float	shape: Shape{1, 64, 8,
// 8}
//	- name: Convolution_1349_0	type: float	shape: Shape{1, 64, 8,
// 8}
// Output:
//	- name: Concat_1353_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Concat_float_float_float_cuda_Concat_1353_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  uint32_t inputs_strides[] = {4096, 4096};
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < 8192) {
    uint32_t block_id = tid / 8192;
    uint32_t block_idx = tid % 8192;
    uint32_t output_idx = block_id * 8192 + block_idx;
    if (block_idx < inputs_strides[0]) {
      output0[output_idx] = input0[block_id * inputs_strides[0] + block_idx];
      return;
    }
    block_idx -= inputs_strides[0];
    if (block_idx < inputs_strides[1]) {
      output0[output_idx] = input1[block_id * inputs_strides[1] + block_idx];
      return;
    }
    block_idx -= inputs_strides[1];
  }
}
// Node name:	DepthwiseConv2dNative_1354
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1350_0	type: float	shape: Shape{1, 128, 16, 16}
//	- name: Constant_218_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1354_0	type: float	shape: Shape{1,
// 128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1354_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 128;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 2;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	Add_1355
// Description:	Add
// Input:
//	- name: MaxPool_1351_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1325_0	type: float	shape: Shape{1,
// 128, 8, 8}
// Output:
//	- name: Add_1355_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1355_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Concat_DepthwiseConv2dNative_Add_Add_122(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *output0, float *output1,
    float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Concat_float_float_float_cuda_Concat_1353_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_1355_block_kernel(
        input4, input5, output2, threadIdx.x, blockIdx.x - 16, NULL);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 47) {
    Add_float_float_float_cuda_Add_1355_block_kernel(
        input6, input7, output3, threadIdx.x, blockIdx.x - 32, NULL);
  } else if ((int)blockIdx.x >= 48 && (int)blockIdx.x <= 111) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1354_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 48, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Concat_DepthwiseConv2dNative_Add_Add_122_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *output0, float *output1,
    float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Concat_DepthwiseConv2dNative_Add_Add_122<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, input7, output0, output1,
                                    output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1198_0	type: float	shape: Shape{1,
// 64, 16, 16}
//	- name: Constant_2431_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_1199_0	type: float	shape: Shape{1,
// 64, 16, 16}
//	- name: Constant_2434_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_1204_0	type: float	shape: Shape{1, 64, 16,
// 16}
//	- name: Convolution_1206_0	type: float	shape: Shape{1, 64, 16,
// 16}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1204<<<dim3(1, 16, 4), dim3(8,
// 1, 16), 0, 0>>>(DepthwiseConv2dNative_1198_0, Constant_2431_0,
// Convolution_1204_0);
// Convolution_float_float_float_cuda_Convolution_1206<<<dim3(1, 16, 4), dim3(8,
// 1, 16), 0, 0>>>(DepthwiseConv2dNative_1199_0, Constant_2434_0,
// Convolution_1206_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_1206 :
// Convolution_float_float_float_cuda_Convolution_1204

// Node name:	Convolution_1204
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1198_0	type: float	shape: Shape{1,
// 64, 16, 16}
//	- name: Constant_2431_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_1204_0	type: float	shape: Shape{1, 64, 16,
// 16}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_1204_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 1, 16);
  const dim3 gridDim(1, 16, 4);
  const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute_local[2];

      compute_local[0] = 0.000000e+00f;
      compute_local[1] = 0.000000e+00f;
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[(((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                  (((int)threadIdx.x) * 2))];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  16)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  17)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8192)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8193)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  32)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  33)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12288)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12289)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  48)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  49)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                (((int)blockIdx.y) * 16)) +
               (((int)threadIdx.x) * 2))] = compute_local[0];
      compute[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.x) * 2)) +
               1)] = compute_local[1];
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_99(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[2048];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_1204_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_1204_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_99_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_99<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
