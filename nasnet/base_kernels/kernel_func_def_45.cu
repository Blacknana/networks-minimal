#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_2766
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2766_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2766(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2766_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2766_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2584
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2584_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2584(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2584_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2584_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2638
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2638_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2638(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2638_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2638_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2976
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2976_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2976(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2976_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2976_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3106
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3106_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3106(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3106_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3106_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3158
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3158_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3158(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3158_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3158_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3018
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3018_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3018(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3018_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3018_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_340
// Description:	Constant
// Input:
// Output:
//	- name: Constant_340_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_340(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_340_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_340_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[6400];
  bin_file.read(tmp_mem, 6400);
  hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2998
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2998_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2998(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2998_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2998_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_90
// Description:	Constant
// Input:
// Output:
//	- name: Constant_90_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_90(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_90_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_90_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2185
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2185_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2185(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2185_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2185_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_134
// Description:	Constant
// Input:
// Output:
//	- name: Constant_134_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_134(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_134_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_134_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[2304];
  bin_file.read(tmp_mem, 2304);
  hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_160
// Description:	Constant
// Input:
// Output:
//	- name: Constant_160_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_160(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_160_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_160_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_857_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_246_0	type: float	shape: Shape{7, 7, 64, 1}
//	- name: Relu_855_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_303_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Relu_856_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_385_0	type: float	shape: Shape{7, 7, 64, 1}
//	- name: Slice_833_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: DepthwiseConv2dNative_862_0	type: float	shape: Shape{1,
// 64, 16, 16}
//	- name: DepthwiseConv2dNative_860_0	type: float	shape: Shape{1,
// 64, 16, 16}
//	- name: DepthwiseConv2dNative_861_0	type: float	shape: Shape{1,
// 64, 16, 16}
//	- name: Relu_838_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_862<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_857_0, Constant_246_0,
// DepthwiseConv2dNative_862_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_860<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_855_0, Constant_303_0,
// DepthwiseConv2dNative_860_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_861<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_856_0, Constant_385_0,
// DepthwiseConv2dNative_861_0); Relu_float_float_cuda_Relu_838<<<dim3(64, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Slice_833_0, Relu_838_0); Deduped function map:
// <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_861 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_862

// Node name:	DepthwiseConv2dNative_862
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_857_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_246_0	type: float	shape: Shape{7, 7, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_862_0	type: float	shape: Shape{1,
// 64, 16, 16}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_862_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 64;
  const int filter_height = 7;
  const int filter_width = 7;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 3;
  const int pad_width = 3;
  const int out_height = 16;
  const int out_width = 16;
  const int out_depth = 64;
  const int num_outputs = 16384;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_860
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_855_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_303_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_860_0	type: float	shape: Shape{1,
// 64, 16, 16}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_860_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 64;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 16;
  const int out_width = 16;
  const int out_depth = 64;
  const int num_outputs = 16384;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	Relu_838
// Description:	Relu
// Input:
//	- name: Slice_833_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_838_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_838_block_kernel(float *input0, float *output0,
                                            int thread_id, int block_id,
                                            char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Relu_48(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *output0, float *output1,
    float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Relu_float_float_cuda_Relu_838_block_kernel(input6, output3, threadIdx.x,
                                                blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 191) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_862_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 64, NULL);
  } else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 319) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_860_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 192, NULL);
  } else if ((int)blockIdx.x >= 320 && (int)blockIdx.x <= 447) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_862_block_kernel(
        input4, input5, output2, threadIdx.x, blockIdx.x - 320, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Relu_48_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *output0, float *output1,
    float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Relu_48<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, output0, output1, output2,
                                    output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1523_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2608_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1524_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2611_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1529_0	type: float	shape: Shape{1, 128, 8,
// 8}
//	- name: Convolution_1531_0	type: float	shape: Shape{1, 128, 8,
// 8}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1529<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1523_0, Constant_2608_0,
// Convolution_1529_0);
// Convolution_float_float_float_cuda_Convolution_1531<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1524_0, Constant_2611_0,
// Convolution_1531_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_1531 :
// Convolution_float_float_float_cuda_Convolution_1529

// Node name:	Convolution_1529
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1523_0	type: float	shape: Shape{1,
// 128, 8, 8}
//	- name: Constant_2608_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1529_0	type: float	shape: Shape{1, 128, 8,
// 8}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_1529_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute_local[1];

      compute_local[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] = compute_local[0];
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_146(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_1529_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_1529_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_146_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_146<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
