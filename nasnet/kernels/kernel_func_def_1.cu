#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
// Node name:	Constant_196
// Description:	Constant
// Input:
// Output:
//	- name: Constant_196_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_196(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_196_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_196_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2392
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2392_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2392(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2392_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2392_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_193
// Description:	Constant
// Input:
// Output:
//	- name: Constant_193_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_193(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_193_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_193_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2731
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2731_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2731(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2731_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2731_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2482
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2482_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2482(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2482_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2482_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_183
// Description:	Constant
// Input:
// Output:
//	- name: Constant_183_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_183(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_183_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_183_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2785
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2785_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2785(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2785_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2785_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2739
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2739_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2739(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2739_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2739_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2113
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2113_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2113(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2113_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2113_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2872
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2872_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2872(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2872_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2872_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2419
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2419_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2419(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2419_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2419_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_871_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2254_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_872_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2257_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Convolution_879_0	type: float	shape: Shape{1, 32, 32,
//32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_877<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_871_0, Constant_2254_0,
// Convolution_877_0);
// Convolution_float_float_float_cuda_Convolution_879<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_872_0, Constant_2257_0,
// Convolution_879_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_879 :
// Convolution_float_float_float_cuda_Convolution_877

// Node name:	Convolution_877
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_871_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2254_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32,
//32}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_877_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 2, 8);
  const dim3 gridDim(2, 16, 2);
  const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute_local[2];

      for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
        compute_local[ff_c_init] = 0.000000e+00f;
      }
      for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) +
               (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                      (((int)threadIdx.y) * 1024)) +
                     (((int)blockIdx.y) * 64)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      4) *
                     32)) +
                   (((int)blockIdx.x) * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   15))];
        }
        input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       ((int)threadIdx.x))] =
            input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                      (((int)threadIdx.y) * 32)) +
                     (rc_outer * 16)) +
                    ((int)threadIdx.x))];
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[ff_c] =
                (compute_local[ff_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.y) * 16)) +
                      ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) +
                                 rc_inner)]));
          }
        }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2;
           ++ff_inner_inner_inner) {
        compute[(
            ((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) +
                (ff_inner_inner_inner * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (((int)threadIdx.y) * 32)) +
             (((int)blockIdx.x) * 16)) +
            ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
      }
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_52(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[3072];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_877_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_877_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_52_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_52<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_742_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2777_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_744_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2744_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_2836_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_746_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Relu_739_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_456_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Relu_740_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_357_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: BatchNormInference_749_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Add_756_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_747_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_748_0	type: float	shape: Shape{1,
//32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2175<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_742_0, Constant_2777_0, BatchNormInference_749_0);
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_16<<<dim3(64, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_744_0, Constant_2744_0,
// Convolution_746_0, Constant_2836_0, Add_756_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_747<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_739_0, Constant_456_0,
// DepthwiseConv2dNative_747_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_748<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_740_0, Constant_357_0,
// DepthwiseConv2dNative_748_0); Deduped function map: <src_function_name :
// deduped_function_name>

// Node name:	Add_2175
// Description:	Add
// Input:
//	- name: Convolution_742_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2777_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: BatchNormInference_749_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_2175_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_744_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2744_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_746_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2836_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_756_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2178<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_744_0, Constant_2744_0, BatchNormInference_750_0);
// Add_float_float_float_cuda_Add_2181<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_746_0, Constant_2836_0, BatchNormInference_751_0);
// Add_float_float_float_cuda_Add_756<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_751_0, BatchNormInference_750_0, Add_756_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_16_block_kernel(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(input2[tid], input3[tid]);
  float temp2 = add(temp1, temp0);
  output0[tid] = temp2;
}
// Node name:	DepthwiseConv2dNative_747
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_739_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_456_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_747_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_747_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_748
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_740_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_357_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_748_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_748_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_fused_kernel_DepthwiseConv2dNative_DepthwiseConv2dNative_33(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Add_float_float_float_cuda_Add_2175_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_16_block_kernel(
        input2, input3, input5, input4, output1, threadIdx.x, blockIdx.x - 64,
        NULL);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 383) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_747_block_kernel(
        input6, input7, output2, threadIdx.x, blockIdx.x - 128, NULL);
  } else if ((int)blockIdx.x >= 384 && (int)blockIdx.x <= 639) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_748_block_kernel(
        input8, input9, output3, threadIdx.x, blockIdx.x - 384, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_fused_kernel_DepthwiseConv2dNative_DepthwiseConv2dNative_33_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_fused_kernel_DepthwiseConv2dNative_DepthwiseConv2dNative_33<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, input7, input8, input9,
                                    output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: AvgPool_587_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_488_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Relu_588_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_352_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Constant_289_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Relu_607_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_155_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Relu_605_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_260_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Relu_606_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_153_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: Add_595_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_596_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_597_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_612_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_610_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_611_0	type: float	shape: Shape{1,
//32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_595<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(AvgPool_587_0, BatchNormInference_488_0, Add_595_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_588_0, Constant_352_0,
// DepthwiseConv2dNative_596_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_597<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_588_0, Constant_289_0,
// DepthwiseConv2dNative_597_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_612<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_607_0, Constant_155_0,
// DepthwiseConv2dNative_612_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_610<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_605_0, Constant_260_0,
// DepthwiseConv2dNative_610_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_611<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_606_0, Constant_153_0,
// DepthwiseConv2dNative_611_0); Deduped function map: <src_function_name :
// deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_612 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_597
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_610 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_611 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596

// Node name:	Add_595
// Description:	Add
// Input:
//	- name: AvgPool_587_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_488_0	type: float	shape: Shape{1,
//32, 32, 32}
// Output:
//	- name: Add_595_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ void Add_float_float_float_cuda_Add_595_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	DepthwiseConv2dNative_596
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_588_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_352_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_596_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_597
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_588_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_289_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_597_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_597_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_13(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Add_float_float_float_cuda_Add_595_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 319) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, NULL);
  } else if ((int)blockIdx.x >= 320 && (int)blockIdx.x <= 575) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_597_block_kernel(
        input2, input4, output2, threadIdx.x, blockIdx.x - 320, NULL);
  } else if ((int)blockIdx.x >= 576 && (int)blockIdx.x <= 831) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_597_block_kernel(
        input5, input6, output3, threadIdx.x, blockIdx.x - 576, NULL);
  } else if ((int)blockIdx.x >= 832 && (int)blockIdx.x <= 1087) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596_block_kernel(
        input7, input8, output4, threadIdx.x, blockIdx.x - 832, NULL);
  } else if ((int)blockIdx.x >= 1088 && (int)blockIdx.x <= 1343) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_596_block_kernel(
        input9, input10, output5, threadIdx.x, blockIdx.x - 1088, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_13_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_13<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4, output5);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1124_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_324_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Relu_1125_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_387_0	type: float	shape: Shape{3, 3, 64, 1}
//	- name: Convolution_1136_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Constant_2899_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2896_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1132_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Convolution_1134_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Constant_2900_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: DepthwiseConv2dNative_1129_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: DepthwiseConv2dNative_1130_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: BatchNormInference_1143_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Add_1146_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1129<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1124_0, Constant_324_0,
// DepthwiseConv2dNative_1129_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1130<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1125_0, Constant_387_0,
// DepthwiseConv2dNative_1130_0); Add_float_float_float_cuda_Add_2394<<<dim3(32,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1136_0, Constant_2899_0,
// BatchNormInference_1143_0);
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_37<<<dim3(32, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1132_0, Constant_2896_0,
// Convolution_1134_0, Constant_2900_0, Add_1146_0); Deduped function map:
// <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_1129
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1124_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_324_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1129_0	type: float	shape: Shape{1,
//64, 16, 16}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1129_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 64;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 16;
  const int out_width = 16;
  const int out_depth = 64;
  const int num_outputs = 16384;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_1130
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1125_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_387_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1130_0	type: float	shape: Shape{1,
//64, 16, 16}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1130_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 64;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 16;
  const int out_width = 16;
  const int out_depth = 64;
  const int num_outputs = 16384;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	Add_2394
// Description:	Add
// Input:
//	- name: Convolution_1136_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Constant_2899_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: BatchNormInference_1143_0	type: float	shape: Shape{1,
//64, 16, 16}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_2394_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1132_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Constant_2896_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1134_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Constant_2900_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1146_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Add_float_float_float_cuda_Add_2388<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1132_0, Constant_2896_0, BatchNormInference_1141_0);
// Add_float_float_float_cuda_Add_2391<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1134_0, Constant_2900_0, BatchNormInference_1142_0);
// Add_float_float_float_cuda_Add_1146<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1141_0, BatchNormInference_1142_0, Add_1146_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_37_block_kernel(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(input2[tid], input3[tid]);
  float temp2 = add(temp0, temp1);
  output0[tid] = temp2;
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_fused_kernel_89(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_2394_block_kernel(
        input4, input5, output2, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 63) {
    FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_37_block_kernel(
        input7, input6, input8, input9, output3, threadIdx.x, blockIdx.x - 32,
        NULL);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 191) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1129_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 64, NULL);
  } else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 319) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1130_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 192, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_fused_kernel_89_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_fused_kernel_89<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, input7, input8, input9,
                                    output0, output1, output2, output3);
}
