#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
// Node name:	Constant_94
// Description:	Constant
// Input:
// Output:
//	- name: Constant_94_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_94(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_94_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_94_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2110
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2110_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2110(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2110_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2110_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2590
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2590_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2590(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2590_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2590_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_218
// Description:	Constant
// Input:
// Output:
//	- name: Constant_218_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_218(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_218_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_218_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_223
// Description:	Constant
// Input:
// Output:
//	- name: Constant_223_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_223(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_223_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_223_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2984
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2984_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2984(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2984_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2984_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2335
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2335_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2335(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2335_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2335_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3008
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3008_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3008(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3008_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3008_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2916
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2916_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2916(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2916_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2916_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2778
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2778_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2778(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2778_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2778_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2740
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2740_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2740(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2740_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2740_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1745_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2731_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1746_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2734_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1748_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Convolution_1750_0	type: float	shape: Shape{1, 128, 8,
//8}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1748<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1745_0, Constant_2731_0,
// Convolution_1748_0);
// Convolution_float_float_float_cuda_Convolution_1750<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1746_0, Constant_2734_0,
// Convolution_1750_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_1750 :
// Convolution_float_float_float_cuda_Convolution_1748

// Node name:	Convolution_1748
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1745_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2731_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1748_0	type: float	shape: Shape{1, 128, 8,
//8}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_1748_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute_local[1];

      compute_local[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] = compute_local[0];
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_178(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_1748_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_1748_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_178_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_178<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
// Node name:	Constant_2716
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2716_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2716(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2716_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2716_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_558_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2074_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_559_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2077_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_564_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Convolution_566_0	type: float	shape: Shape{1, 32, 32,
//32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_564<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_558_0, Constant_2074_0,
// Convolution_564_0);
// Convolution_float_float_float_cuda_Convolution_566<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_559_0, Constant_2077_0,
// Convolution_566_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_566 :
// Convolution_float_float_float_cuda_Convolution_564

// Node name:	Convolution_564
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_558_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2074_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_564_0	type: float	shape: Shape{1, 32, 32,
//32}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_564_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 2, 8);
  const dim3 gridDim(2, 16, 2);
  const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute_local[2];

      for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
        compute_local[ff_c_init] = 0.000000e+00f;
      }
      for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) +
               (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                      (((int)threadIdx.y) * 1024)) +
                     (((int)blockIdx.y) * 64)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      4) *
                     32)) +
                   (((int)blockIdx.x) * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   15))];
        }
        input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       ((int)threadIdx.x))] =
            input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                      (((int)threadIdx.y) * 32)) +
                     (rc_outer * 16)) +
                    ((int)threadIdx.x))];
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[ff_c] =
                (compute_local[ff_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.y) * 16)) +
                      ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) +
                                 rc_inner)]));
          }
        }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2;
           ++ff_inner_inner_inner) {
        compute[(
            ((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) +
                (ff_inner_inner_inner * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (((int)threadIdx.y) * 32)) +
             (((int)blockIdx.x) * 16)) +
            ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
      }
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_7(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[3072];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_564_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_564_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_7_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_7<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
// Node name:	AvgPool_509
// Description:	AvgPool
// Input:
//	- name: Slice_500_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: AvgPool_509_0	type: float	shape: Shape{1, 32, 32, 32}
void AvgPool_float_float_cuda_lib_AvgPool_509(hipdnnHandle_t cudnn_handle,
                                              float *input0, float *output0) {
  hipdnnTensorDescriptor_t input_desc;
  CUDNN_SAFE_CALL(hipdnnCreateTensorDescriptor(&input_desc));
  CUDNN_SAFE_CALL(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW,
                                             HIPDNN_DATA_FLOAT, 1, 32, 32, 32));
  hipdnnTensorDescriptor_t output_desc;
  CUDNN_SAFE_CALL(hipdnnCreateTensorDescriptor(&output_desc));
  CUDNN_SAFE_CALL(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW,
                                             HIPDNN_DATA_FLOAT, 1, 32, 32, 32));
  hipdnnPoolingDescriptor_t desc;
  hipdnnCreatePoolingDescriptor(&desc);
  CUDNN_SAFE_CALL(hipdnnSetPooling2dDescriptor(
      desc, HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,
      HIPDNN_NOT_PROPAGATE_NAN, 3, 3, 1, 1, 1, 1));
  const float alpha = 1.0;
  const float beta = 0.0;
  CUDNN_SAFE_CALL(hipdnnPoolingForward(cudnn_handle, desc, &alpha, input_desc,
                                      input0, &beta, output_desc, output0));
  CUDNN_SAFE_CALL(hipdnnDestroyTensorDescriptor(input_desc));
  CUDNN_SAFE_CALL(hipdnnDestroyTensorDescriptor(output_desc));
  CUDNN_SAFE_CALL(hipdnnDestroyPoolingDescriptor(desc));
}

// Node name:	AvgPool_509
// Description:	AvgPool
// Input:
//	- name: Slice_500_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: AvgPool_509_0	type: float	shape: Shape{1, 32, 32, 32}
// 3, 1, 1(kernelH, pad, stride)
// grid(128,1,1) block(256,1,1)
__device__ void operator_avg_pool_h_32_32_32_3x3_1(const float *input,
                                                   float *output,
                                                   int blockidx) {

  const int pooled_height = 32;
  const int pooled_width = 32;
  const int nthreads = 32768;
  int index = blockidx * 256 + threadIdx.x;

  if (index < nthreads) {
    const int kChannels = 32;
    const int kHeight = 32;
    const int kWidth = 32;
    const int kKernelH = 3;
    const int kKernelW = 3;
    const int kPadH = 1;
    const int kPadW = 1;
    const int kStrideH = 1;
    const int kStrideW = 1;

    // output location
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % kChannels;
    const int n = index / pooled_width / pooled_height / kChannels;

    // pooled range
    int hstart = ph * kStrideH - kPadH;
    int wstart = pw * kStrideW - kPadW;
    const int hend = fminf(hstart + kKernelH, kHeight);
    const int wend = fminf(wstart + kKernelW, kWidth);
    hstart = fmaxf(hstart, 0);
    wstart = fmaxf(wstart, 0);

    float avgval = 0.0f;
    int slice_offset = (n * kChannels + c) * kHeight * kWidth;
#pragma unroll 4
    for (int h = hstart; h < hend; ++h) {
#pragma unroll 4
      for (int w = wstart; w < wend; ++w) {
        avgval = (input[slice_offset + h * kWidth + w]) /
                     ((hend - hstart) * (wend - wstart)) +
                 avgval;
      }
    }

    // output
    output[index] = avgval;
  }
}

extern "C" __global__ void BlockFusionKernel_2_AvgPool_509(float *input0,
                                                           float *input1,
                                                           float *output0,
                                                           float *output1) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 127) {
    operator_avg_pool_h_32_32_32_3x3_1(input0, output0, blockIdx.x);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 255) {
    operator_avg_pool_h_32_32_32_3x3_1(input1, output1, blockIdx.x - 128);
  }
}

extern void BlockFusionKernel_2_AvgPool_509_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *output0, float *output1) {
  BlockFusionKernel_2_AvgPool_509<<<grids, blocks, mem, stream>>>(
      input0, input1, output0, output1);
}
