#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_2362
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2362_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2362(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2362_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2362_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2805
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2805_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2805(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2805_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2805_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_74
// Description:	Constant
// Input:
// Output:
//	- name: Constant_74_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_74(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_74_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_74_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3200];
  bin_file.read(tmp_mem, 3200);
  hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_67
// Description:	Constant
// Input:
// Output:
//	- name: Constant_67_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_67(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_67_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_67_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3170
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3170_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3170(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3170_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3170_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_457
// Description:	Constant
// Input:
// Output:
//	- name: Constant_457_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_457(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_457_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_457_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_307
// Description:	Constant
// Input:
// Output:
//	- name: Constant_307_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_307(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_307_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_307_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[6400];
  bin_file.read(tmp_mem, 6400);
  hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3032
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3032_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3032(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3032_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3032_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_121
// Description:	Constant
// Input:
// Output:
//	- name: Constant_121_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_121(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_121_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_121_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[6400];
  bin_file.read(tmp_mem, 6400);
  hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Convolution_466
// Description:	Convolution
// Input:
//	- name: Reshape_464_0	type: float	shape: Shape{1, 3, 32, 32}
//	- name: Constant_2011_0	type: float	shape: Shape{96, 3, 3, 3}
// Output:
//	- name: Convolution_466_0	type: float	shape: Shape{1, 96, 32,
//32}
extern "C" __global__ void
Convolution_float_float_float_cuda_Convolution_466(float *input0, float *input1,
                                                   float *output0) {
  __shared__ float pad_temp_shared[408];
  __shared__ float input1_shared[864];
  {
    float *compute = output0;
    {
      float compute_local[4];

#pragma unroll
      for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
        compute_local[yy_c_init] = 0.000000e+00f;
        compute_local[(yy_c_init + 2)] = 0.000000e+00f;
      }
      if (((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) < 408) {
        if (((int)threadIdx.x) < 26) {
          pad_temp_shared[((((int)threadIdx.z) * 26) + ((int)threadIdx.x))] =
              (((((1 <=
                   ((((int)blockIdx.y) * 2) +
                    ((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 136) /
                     34))) &&
                  (((((int)blockIdx.y) * 2) +
                    ((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 136) /
                     34)) < 33)) &&
                 (1 <=
                  (((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 34))) &&
                ((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) % 34) < 33))
                   ? input0[(
                         (((((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) /
                              136) *
                             1024) +
                            (((int)blockIdx.y) * 64)) +
                           (((((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) %
                              136) /
                             34) *
                            32)) +
                          (((((int)threadIdx.z) * 26) + ((int)threadIdx.x)) %
                           34)) -
                         33)]
                   : 0.000000e+00f);
        }
      }
#pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
           ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
           ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        if (((((int)threadIdx.z) * 2) +
             (((((int)threadIdx.x) * 2) +
               ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) /
              27)) < 32) {
          if (((((int)threadIdx.z) * 6) +
               (((((int)threadIdx.x) * 2) +
                 ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) /
                9)) < 96) {
            if (((((int)threadIdx.z) * 18) +
                 (((((int)threadIdx.x) * 2) +
                   ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) /
                  3)) < 288) {
              if ((((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 2)) +
                   ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 864) {
                if (((((int)threadIdx.x) * 2) +
                     ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) <
                    54) {
                  input1_shared[(
                      ((((int)threadIdx.z) * 54) + (((int)threadIdx.x) * 2)) +
                      ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
                      input1[(
                          (((((int)blockIdx.z) * 864) +
                            (((int)threadIdx.z) * 54)) +
                           (((int)threadIdx.x) * 2)) +
                          ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)];
                }
              }
            }
          }
        }
      }
      __syncthreads();
#pragma unroll
      for (int rc_inner = 0; rc_inner < 3; ++rc_inner) {
#pragma unroll
        for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
#pragma unroll
          for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
#pragma unroll
            for (int yy_c = 0; yy_c < 2; ++yy_c) {
              compute_local[yy_c] =
                  (compute_local[yy_c] +
                   (pad_temp_shared[(
                        ((((rc_inner * 136) + (yy_c * 34)) + (ry_inner * 34)) +
                         ((int)threadIdx.x)) +
                        rx_inner)] *
                    input1_shared[(
                        (((((int)threadIdx.z) * 27) + (rc_inner * 9)) +
                         (ry_inner * 3)) +
                        rx_inner)]));
              compute_local[(yy_c + 2)] =
                  (compute_local[(yy_c + 2)] +
                   (pad_temp_shared[(
                        ((((rc_inner * 136) + (yy_c * 34)) + (ry_inner * 34)) +
                         ((int)threadIdx.x)) +
                        rx_inner)] *
                    input1_shared[(
                        ((((((int)threadIdx.z) * 27) + (rc_inner * 9)) +
                          (ry_inner * 3)) +
                         rx_inner) +
                        432)]));
            }
          }
        }
      }
#pragma unroll
      for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2;
           ++yy_inner_inner_inner) {
        compute[(((((((int)blockIdx.z) * 32768) + (((int)threadIdx.z) * 1024)) +
                   (((int)blockIdx.y) * 64)) +
                  (yy_inner_inner_inner * 32)) +
                 ((int)threadIdx.x))] = compute_local[yy_inner_inner_inner];
        compute[(
            (((((((int)blockIdx.z) * 32768) + (((int)threadIdx.z) * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (yy_inner_inner_inner * 32)) +
             ((int)threadIdx.x)) +
            16384)] = compute_local[(yy_inner_inner_inner + 2)];
      }
    }
  }
}
extern void Convolution_float_float_float_cuda_Convolution_466_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *output0) {
  Convolution_float_float_float_cuda_Convolution_466<<<grids, blocks, mem,
                                                       stream>>>(input0, input1,
                                                                 output0);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1509_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_99_0	type: float	shape: Shape{3, 3, 128, 1}
//	- name: Relu_1507_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_111_0	type: float	shape: Shape{3, 3, 128, 1}
//	- name: Relu_1508_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_252_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: AvgPool_1489_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1417_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Relu_1490_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_259_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Constant_223_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1514_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1512_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1513_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Add_1497_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1498_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1499_0	type: float	shape: Shape{1,
//128, 8, 8}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1509_0, Constant_99_0,
// DepthwiseConv2dNative_1514_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1512<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1507_0, Constant_111_0,
// DepthwiseConv2dNative_1512_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1513<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1508_0, Constant_252_0,
// DepthwiseConv2dNative_1513_0); Add_float_float_float_cuda_Add_1497<<<dim3(16,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1489_0, BatchNormInference_1417_0,
// Add_1497_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1498<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1490_0, Constant_259_0,
// DepthwiseConv2dNative_1498_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1499<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1490_0, Constant_223_0,
// DepthwiseConv2dNative_1499_0); Deduped function map: <src_function_name :
// deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1512 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1498 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1513
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1499 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514

// Node name:	DepthwiseConv2dNative_1514
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1509_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_99_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1514_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_1513
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1508_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_252_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1513_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1513_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	Add_1497
// Description:	Add
// Input:
//	- name: AvgPool_1489_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1417_0	type: float	shape: Shape{1,
//128, 8, 8}
// Output:
//	- name: Add_1497_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1497_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_143(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Add_float_float_float_cuda_Add_1497_block_kernel(
        input6, input7, output3, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 79) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 16, NULL);
  } else if ((int)blockIdx.x >= 80 && (int)blockIdx.x <= 143) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 80, NULL);
  } else if ((int)blockIdx.x >= 144 && (int)blockIdx.x <= 207) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1513_block_kernel(
        input4, input5, output2, threadIdx.x, blockIdx.x - 144, NULL);
  } else if ((int)blockIdx.x >= 208 && (int)blockIdx.x <= 271) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1513_block_kernel(
        input8, input9, output4, threadIdx.x, blockIdx.x - 208, NULL);
  } else if ((int)blockIdx.x >= 272 && (int)blockIdx.x <= 335) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1514_block_kernel(
        input8, input10, output5, threadIdx.x, blockIdx.x - 272, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_143_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_143<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4, output5);
}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_1545_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1550_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2620_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3150_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1552_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2626_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3154_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1547_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1551_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2623_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3152_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1549_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1572_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1574_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1553_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1573_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Relu_float_float_cuda_Relu_1549<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Slice_1545_0, Relu_1549_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1550_0, Constant_2620_0,
// Constant_3150_0, Relu_1572_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3153<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1552_0, Constant_2626_0,
// Constant_3154_0, Relu_1574_0); Add_float_float_float_cuda_Add_1553<<<dim3(16,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1547_0, AvgPool_1547_0, Add_1553_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3151<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1551_0, Constant_2623_0,
// Constant_3152_0, Relu_1573_0); Deduped function map: <src_function_name :
// deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3153 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3151 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149

// Node name:	Relu_1549
// Description:	Relu
// Input:
//	- name: Slice_1545_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1549_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_1549_block_kernel(float *input0, float *output0,
                                             int thread_id, int block_id,
                                             char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Matched_Pattern_3149
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1550_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2620_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3150_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1572_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[1];

      compute1[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
// Node name:	Add_1553
// Description:	Add
// Input:
//	- name: AvgPool_1547_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1547_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1553_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1553_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_151(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Relu_float_float_cuda_Relu_1549_block_kernel(input0, output0, threadIdx.x,
                                                 blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_1553_block_kernel(
        input7, input7, output3, threadIdx.x, blockIdx.x - 16, shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149_block_kernel(
        input1, input2, input3, output1, threadIdx.x, blockIdx.x - 32,
        shared_buffer);
  } else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149_block_kernel(
        input4, input5, input6, output2, threadIdx.x, blockIdx.x - 96,
        shared_buffer);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3149_block_kernel(
        input8, input9, input10, output4, threadIdx.x, blockIdx.x - 160,
        shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_151_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_151<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_767_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Relu_771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_356_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_68_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Constant_235_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: Slice_769_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_775_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_777_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_776_0	type: float	shape: Shape{1,
//32, 32, 32}
// Fused functions:
// Slice_float_float_cuda_Slice_769<<<dim3(512, 1, 1), dim3(64, 1, 1), 0,
// 0>>>(BatchNormInference_767_0, Slice_769_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_775<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_771_0, Constant_356_0,
// DepthwiseConv2dNative_775_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_771_0, Constant_68_0,
// DepthwiseConv2dNative_777_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_776<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_771_0, Constant_235_0,
// DepthwiseConv2dNative_776_0); Deduped function map: <src_function_name :
// deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_776 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777

// Node name:	Slice_769
// Description:	Slice
// Input:
//	- name: BatchNormInference_767_0	type: float	shape: Shape{1,
//32, 32, 32}
// Output:
//	- name: Slice_769_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ void
Slice_float_float_cuda_Slice_769_block_kernel(float *input0, float *output0,
                                              int thread_id, int block_id,
                                              char *shared_buffer) {
  if (thread_id >= 64) {
    return;
  }
  const dim3 blockDim(64, 1, 1);
  const dim3 gridDim(512, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < 32768) {
    uint32_t input_strides[] = {32768, 1024, 32, 1};
    uint32_t output_strides[] = {32768, 1024, 32, 1};
    uint32_t lower_bounds[] = {0, 0, 0, 0};
    uint32_t slice_strides[] = {1, 1, 1, 1};
    uint32_t input_idx = 0;
    uint32_t output_idx = tid;
    input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) +
                  lower_bounds[0]) *
                 input_strides[0];
    output_idx %= output_strides[0];
    input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) +
                  lower_bounds[1]) *
                 input_strides[1];
    output_idx %= output_strides[1];
    input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) +
                  lower_bounds[2]) *
                 input_strides[2];
    output_idx %= output_strides[2];
    input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) +
                  lower_bounds[3]) *
                 input_strides[3];
    output0[tid] = input0[input_idx];
  }
}
// Node name:	DepthwiseConv2dNative_775
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_356_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_775_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_775_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_777
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_68_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_777_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
__device__ __forceinline__ static void
fused2_DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(
    float *input0, float *input1, float *input2, float *output0, float *output1,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    S sum2 = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
          sum2 += static_cast<S>(__ldg(input + input_offset)) *
                  static_cast<S>(__ldg(input2 + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
            sum2 += static_cast<S>(__ldg(input + input_offset)) *
                    static_cast<S>(__ldg(input2 + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
    output1[thread_id] = static_cast<S>(sum2);
  }
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_38(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *output0, float *output1, float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 255) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_775_block_kernel(
        input1, input2, output1, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 511) {
    fused2_DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(
        input1, input3, input4, output2, output3, threadIdx.x, blockIdx.x - 256,
        NULL);
  }
  // else if((int)blockIdx.x >= 512 && (int)blockIdx.x <= 767)
  //{
  //    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_777_block_kernel(input1,
  //    input4, output3, threadIdx.x, blockIdx.x - 512, NULL);
  //}
  else if ((int)blockIdx.x >= 768 - 256 && (int)blockIdx.x <= 1279 - 256) {
    Slice_float_float_cuda_Slice_769_block_kernel(input0, output0, threadIdx.x,
                                                  blockIdx.x - 768 + 256, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_38_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *output0, float *output1, float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_38<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_863_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_463_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Relu_864_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_94_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_871_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_872_0	type: float	shape: Shape{1,
//32, 32, 32}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_871<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_863_0, Constant_463_0,
// DepthwiseConv2dNative_871_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_872<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_864_0, Constant_94_0,
// DepthwiseConv2dNative_872_0); Deduped function map: <src_function_name :
// deduped_function_name>

// Node name:	DepthwiseConv2dNative_871
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_863_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_463_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_871_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_871_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_872
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_864_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_94_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_872_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_872_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_51(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 255) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_871_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 511) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_872_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 256, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_51_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_51<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2869_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1389_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2931_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1391_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Convolution_1393_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2746_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1405_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_441_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: Add_1401_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1398_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1406_0	type: float	shape: Shape{1,
//128, 8, 8}
// Fused functions:
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_50<<<dim3(16, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1389_0, Constant_2869_0,
// Convolution_1391_0, Constant_2931_0, Add_1401_0);
// Add_float_float_float_cuda_Add_2535<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1393_0, Constant_2746_0, BatchNormInference_1398_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1406<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1405_0, Constant_441_0,
// DepthwiseConv2dNative_1406_0); Deduped function map: <src_function_name :
// deduped_function_name>

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1389_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2869_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1391_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2931_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1401_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2529<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1389_0, Constant_2869_0, BatchNormInference_1396_0);
// Add_float_float_float_cuda_Add_2532<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1391_0, Constant_2931_0, BatchNormInference_1397_0);
// Add_float_float_float_cuda_Add_1401<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1396_0, BatchNormInference_1397_0, Add_1401_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_50_block_kernel(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(input2[tid], input3[tid]);
  float temp2 = add(temp0, temp1);
  output0[tid] = temp2;
}
// Node name:	Add_2535
// Description:	Add
// Input:
//	- name: Convolution_1393_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2746_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1398_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_2535_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	DepthwiseConv2dNative_1406
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1405_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_441_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1406_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1406_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_fused_kernel_Add_DepthwiseConv2dNative_129(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *output0, float *output1,
    float *output2) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_50_block_kernel(
        input1, input0, input3, input2, output0, threadIdx.x, blockIdx.x - 0,
        NULL);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_2535_block_kernel(
        input4, input5, output1, threadIdx.x, blockIdx.x - 16, NULL);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1406_block_kernel(
        input6, input7, output2, threadIdx.x, blockIdx.x - 32, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_fused_kernel_Add_DepthwiseConv2dNative_129_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *output0, float *output1,
    float *output2) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_fused_kernel_Add_DepthwiseConv2dNative_129<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, input7, output0, output1,
                                    output2);
}
