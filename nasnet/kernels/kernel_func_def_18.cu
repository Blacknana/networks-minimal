#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
// Node name:	Constant_192
// Description:	Constant
// Input:
// Output:
//	- name: Constant_192_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_192(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_192_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_192_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2524
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2524_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2524(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2524_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2524_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3174
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3174_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3174(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3174_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3174_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_349
// Description:	Constant
// Input:
// Output:
//	- name: Constant_349_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_349(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_349_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_349_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[2304];
  bin_file.read(tmp_mem, 2304);
  hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_462
// Description:	Constant
// Input:
// Output:
//	- name: Constant_462_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_462(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_462_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_462_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[2304];
  bin_file.read(tmp_mem, 2304);
  hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_324
// Description:	Constant
// Input:
// Output:
//	- name: Constant_324_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_324(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_324_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_324_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[6400];
  bin_file.read(tmp_mem, 6400);
  hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3090
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3090_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3090(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3090_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3090_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3066
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3066_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3066(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3066_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3066_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2775
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2775_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2775(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2775_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2775_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_355
// Description:	Constant
// Input:
// Output:
//	- name: Constant_355_0	type: float	shape: Shape{3, 3, 96, 1}
void Constant_float_cuda_Constant_355(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_355_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_355_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3456];
  bin_file.read(tmp_mem, 3456);
  hipMemcpyAsync(output0, tmp_mem, 3456, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_112
// Description:	Constant
// Input:
// Output:
//	- name: Constant_112_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_112(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_112_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_112_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Convolution_957
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_955_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2290_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_957_0	type: float	shape: Shape{1, 64, 16,
//16}
extern "C" __global__ void
Convolution_float_float_float_cuda_Convolution_957(float *input0, float *input1,
                                                   float *output0) {
  __shared__ float pad_temp_shared[256];
  __shared__ float input1_shared[256];
  {
    float *compute = output0;
    {
      float compute_local[2];

      compute_local[0] = 0.000000e+00f;
      compute_local[1] = 0.000000e+00f;
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[(((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                  (((int)threadIdx.x) * 2))];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  16)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  17)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8192)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8193)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  32)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  33)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12288)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12289)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  48)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  49)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                (((int)blockIdx.y) * 16)) +
               (((int)threadIdx.x) * 2))] = compute_local[0];
      compute[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.x) * 2)) +
               1)] = compute_local[1];
    }
  }
}
extern void Convolution_float_float_float_cuda_Convolution_957_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *output0) {
  Convolution_float_float_float_cuda_Convolution_957<<<grids, blocks, mem,
                                                       stream>>>(input0, input1,
                                                                 output0);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2847_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: BatchNormInference_817_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2860_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_879_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Slice_833_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_882_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Add_883_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_23<<<dim3(64, 1, 1),
// dim3(512, 1, 1), 0, 0>>>(Convolution_877_0, Constant_2847_0,
// BatchNormInference_817_0, Add_882_0);
// FusedKernel_float_float_float_float_cuda_Add_Add_24<<<dim3(64, 1, 1),
// dim3(512, 1, 1), 0, 0>>>(Convolution_879_0, Constant_2860_0, Slice_833_0,
// Add_883_0); Deduped function map: <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_24 :
// FusedKernel_float_float_float_float_cuda_Add_Add_23

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_877_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2847_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_817_0	type: float	shape: Shape{1,
//32, 32, 32}
// Output:
//	- name: Add_882_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2256<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_877_0, Constant_2847_0, BatchNormInference_880_0);
// Add_float_float_float_cuda_Add_882<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_880_0, BatchNormInference_817_0, Add_882_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_cuda_Add_Add_23_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(temp0, input2[tid]);
  output0[tid] = temp1;
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_53(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *output0, float *output1) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    FusedKernel_float_float_float_float_cuda_Add_Add_23_block_kernel(
        input1, input0, input2, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    FusedKernel_float_float_float_float_cuda_Add_Add_23_block_kernel(
        input4, input3, input5, output1, threadIdx.x, blockIdx.x - 64, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_53_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *output0, float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_53<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: AvgPool_1337_0	type: float	shape: Shape{1, 384, 8, 8}
//	- name: Constant_1935_0	type: float	shape: Shape{64, 384, 1, 1}
//	- name: Pad_1339_0	type: float	shape: Shape{1, 384, 17, 17}
//	- name: Convolution_1341_0	type: float	shape: Shape{1, 128, 16,
//16}
//	- name: Constant_2928_0	type: float	shape: Shape{1, 128, 16, 16}
// Output:
//	- name: Convolution_1343_0	type: float	shape: Shape{1, 64, 8,
//8}
//	- name: Slice_1344_0	type: float	shape: Shape{1, 384, 16, 16}
//	- name: BatchNormInference_1345_0	type: float	shape: Shape{1,
//128, 16, 16}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1343<<<dim3(1, 8, 8), dim3(8,
// 1, 8), 0, 0>>>(AvgPool_1337_0, Constant_1935_0, Convolution_1343_0);
// Slice_float_float_cuda_Slice_1344<<<dim3(1536, 1, 1), dim3(64, 1, 1), 0,
// 0>>>(Pad_1339_0, Slice_1344_0);
// Add_float_float_float_cuda_Add_2511<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1341_0, Constant_2928_0, BatchNormInference_1345_0); Deduped
// function map: <src_function_name : deduped_function_name>

// Node name:	Convolution_1343
// Description:	Convolution
// Input:
//	- name: AvgPool_1337_0	type: float	shape: Shape{1, 384, 8, 8}
//	- name: Constant_1935_0	type: float	shape: Shape{64, 384, 1, 1}
// Output:
//	- name: Convolution_1343_0	type: float	shape: Shape{1, 64, 8,
//8}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_1343_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 64) {
    return;
  }
  const dim3 blockDim(8, 1, 8);
  const dim3 gridDim(1, 8, 8);
  const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 8, block_id / 8);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 512);
  {
    float *compute = output0;
    {
      float compute_local[1];

      compute_local[0] = 0.000000e+00f;
#pragma unroll
      for (int rc_outer = 0; rc_outer < 24; ++rc_outer) {
        __syncthreads();
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              ((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((rc_outer * 1024) + (((int)threadIdx.z) * 128)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      3) *
                     64)) +
                   (((int)blockIdx.y) * 8)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   7))];
        }
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
          input1_shared[(
              ((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input1[(
                  ((((((int)blockIdx.z) * 3072) + (((int)threadIdx.z) * 384)) +
                    (rc_outer * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)];
        }
        __syncthreads();
#pragma unroll
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          compute_local[0] =
              (compute_local[0] +
               (pad_temp_shared[((rc_inner * 8) + ((int)threadIdx.x))] *
                input1_shared[((((int)threadIdx.z) * 16) + rc_inner)]));
        }
      }
      compute[((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                (((int)blockIdx.y) * 8)) +
               ((int)threadIdx.x))] = compute_local[0];
    }
  }
}
// Node name:	Slice_1344
// Description:	Slice
// Input:
//	- name: Pad_1339_0	type: float	shape: Shape{1, 384, 17, 17}
// Output:
//	- name: Slice_1344_0	type: float	shape: Shape{1, 384, 16, 16}
__device__ __forceinline__ void
Slice_float_float_cuda_Slice_1344_block_kernel(float *input0, float *output0,
                                               int thread_id, int block_id,
                                               char *shared_buffer) {
  if (thread_id >= 64) {
    return;
  }
  const dim3 blockDim(64, 1, 1);
  const dim3 gridDim(1536, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < 98304) {
    uint32_t input_strides[] = {110976, 289, 17, 1};
    uint32_t output_strides[] = {98304, 256, 16, 1};
    uint32_t lower_bounds[] = {0, 0, 1, 1};
    uint32_t slice_strides[] = {1, 1, 1, 1};
    uint32_t input_idx = 0;
    uint32_t output_idx = tid;
    input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) +
                  lower_bounds[0]) *
                 input_strides[0];
    output_idx %= output_strides[0];
    input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) +
                  lower_bounds[1]) *
                 input_strides[1];
    output_idx %= output_strides[1];
    input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) +
                  lower_bounds[2]) *
                 input_strides[2];
    output_idx %= output_strides[2];
    input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) +
                  lower_bounds[3]) *
                 input_strides[3];
    output0[tid] = input0[input_idx];
  }
}
// Node name:	Add_2511
// Description:	Add
// Input:
//	- name: Convolution_1341_0	type: float	shape: Shape{1, 128, 16,
//16}
//	- name: Constant_2928_0	type: float	shape: Shape{1, 128, 16, 16}
// Output:
//	- name: BatchNormInference_1345_0	type: float	shape: Shape{1,
//128, 16, 16}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_2511_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Convolution_Slice_Add_120(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *output0, float *output1, float *output2) {

  __shared__ char shared_buffer[1024];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Add_float_float_float_cuda_Add_2511_block_kernel(
        input3, input4, output2, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_1343_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 64, shared_buffer);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 1663) {
    Slice_float_float_cuda_Slice_1344_block_kernel(
        input2, output1, threadIdx.x, blockIdx.x - 128, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Convolution_Slice_Add_120_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *output0, float *output1, float *output2) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_Convolution_Slice_Add_120<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    output0, output1, output2);
}
// Node name:	Constant_333
// Description:	Constant
// Input:
// Output:
//	- name: Constant_333_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_333(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_333_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_333_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
