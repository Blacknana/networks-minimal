#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_2994
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2994_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2994(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2994_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2994_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2910
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2910_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2910(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2910_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2910_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3168
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3168_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3168(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3168_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3168_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3142
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3142_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3142(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3142_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3142_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2686
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2686_0	type: float	shape: Shape{128, 768, 1, 1}
void Constant_float_cuda_Constant_2686(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2686_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2686_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[393216];
  bin_file.read(tmp_mem, 393216);
  hipMemcpyAsync(output0, tmp_mem, 393216, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2798
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2798_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2798(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2798_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2798_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2928
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2928_0	type: float	shape: Shape{1, 128, 16, 16}
void Constant_float_cuda_Constant_2928(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2928_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2928_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2647
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2647_0	type: float	shape: Shape{128, 128, 1, 1}
void Constant_float_cuda_Constant_2647(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2647_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2647_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2374
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2374_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2374(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2374_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2374_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_133
// Description:	Constant
// Input:
// Output:
//	- name: Constant_133_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_133(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_133_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_133_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[2304];
  bin_file.read(tmp_mem, 2304);
  hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_424
// Description:	Constant
// Input:
// Output:
//	- name: Constant_424_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_424(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_424_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_424_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3200];
  bin_file.read(tmp_mem, 3200);
  hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_289
// Description:	Constant
// Input:
// Output:
//	- name: Constant_289_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_289(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_289_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_289_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3200];
  bin_file.read(tmp_mem, 3200);
  hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_889
// Description:	Constant
// Input:
// Output:
//	- name: Constant_889_0	type: float	shape: Shape{}
void Constant_float_cuda_Constant_889(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_889_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_889_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4];
  bin_file.read(tmp_mem, 4);
  hipMemcpyAsync(output0, tmp_mem, 4, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_895_0	type: float	shape: Shape{1, 64, 32, 32}
//	- name: AvgPool_896_0	type: float	shape: Shape{1, 192, 16, 16}
//	- name: Constant_1853_0	type: float	shape: Shape{32, 192, 1, 1}
// Output:
//	- name: Relu_899_0	type: float	shape: Shape{1, 64, 32, 32}
//	- name: Convolution_901_0	type: float	shape: Shape{1, 32, 16,
//16}
// Fused functions:
// Relu_float_float_cuda_Relu_899<<<dim3(128, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Slice_895_0, Relu_899_0);
// Convolution_float_float_float_cuda_Convolution_901<<<dim3(2, 8, 4), dim3(8,
// 2, 8), 0, 0>>>(AvgPool_896_0, Constant_1853_0, Convolution_901_0); Deduped
// function map: <src_function_name : deduped_function_name>

// Node name:	Relu_899
// Description:	Relu
// Input:
//	- name: Slice_895_0	type: float	shape: Shape{1, 64, 32, 32}
// Output:
//	- name: Relu_899_0	type: float	shape: Shape{1, 64, 32, 32}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_899_block_kernel(float *input0, float *output0,
                                            int thread_id, int block_id,
                                            char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Convolution_901
// Description:	Convolution
// Input:
//	- name: AvgPool_896_0	type: float	shape: Shape{1, 192, 16, 16}
//	- name: Constant_1853_0	type: float	shape: Shape{32, 192, 1, 1}
// Output:
//	- name: Convolution_901_0	type: float	shape: Shape{1, 32, 16,
//16}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_901_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(2, 8, 4);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 8, block_id / 16);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 3072);
  {
    float *compute = output0;
    {
      float compute_local[1];

      compute_local[0] = 0.000000e+00f;
#pragma unroll
      for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
        __syncthreads();
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 6;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 48)) +
               (((int)threadIdx.x) * 6)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  (((((((rc_outer * 12288) + (((int)threadIdx.z) * 1536)) +
                       (((int)threadIdx.y) * 768)) +
                      ((((((int)threadIdx.x) * 6) +
                         ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                        4) *
                       256)) +
                     (((int)blockIdx.y) * 32)) +
                    (((((((int)threadIdx.x) * 6) +
                        ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                       15) >>
                      3) *
                     16)) +
                   (((int)blockIdx.x) * 8)) +
                  (((((int)threadIdx.x) * 6) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   7))];
        }
#pragma unroll
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 3;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
          input1_shared[(
              (((((int)threadIdx.z) * 48) + (((int)threadIdx.y) * 24)) +
               (((int)threadIdx.x) * 3)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)] =
              input1[(
                  (((((((int)blockIdx.z) * 1536) + (((int)threadIdx.z) * 192)) +
                     (rc_outer * 48)) +
                    (((int)threadIdx.y) * 24)) +
                   (((int)threadIdx.x) * 3)) +
                  ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1)];
        }
        __syncthreads();
#pragma unroll
        for (int rc_inner = 0; rc_inner < 48; ++rc_inner) {
          compute_local[0] =
              (compute_local[0] +
               (pad_temp_shared[(((rc_inner * 16) + (((int)threadIdx.y) * 8)) +
                                 ((int)threadIdx.x))] *
                input1_shared[((((int)threadIdx.z) * 48) + rc_inner)]));
        }
      }
      compute[((((((((int)blockIdx.z) * 2048) + (((int)threadIdx.z) * 256)) +
                  (((int)blockIdx.y) * 32)) +
                 (((int)threadIdx.y) * 16)) +
                (((int)blockIdx.x) * 8)) +
               ((int)threadIdx.x))] = compute_local[0];
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_cuda_Relu_Convolution_56(
    float *input0, float *input1, float *input2, float *output0,
    float *output1) {

  __shared__ char shared_buffer[4608];

  // if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 127)
  if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 191) {
    // Relu_float_float_cuda_Relu_899_block_kernel(input0, output0, threadIdx.x,
    // blockIdx.x - 64, shared_buffer);
  }
  // else if((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191)
  else if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    // Convolution_float_float_float_cuda_Convolution_901_block_kernel(input1,
    // input2, output1, threadIdx.x, blockIdx.x - 0, shared_buffer);
  }
}

extern "C" __global__ void
Convolution_float_float_float_cuda_Convolution_901(float *input0, float *input1,
                                                   float *output0) {
  __shared__ char shared_buffer[4608];
  Convolution_float_float_float_cuda_Convolution_901_block_kernel(
      input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
}

extern void
Convolution_float_float_float_cuda_Convolution_901_block_kernel_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *output0) {
  Convolution_float_float_float_cuda_Convolution_901<<<grids, blocks, mem,
                                                       stream>>>(input0, input1,
                                                                 output0);
}
extern void
BlockFusionKernel_float_float_float_float_float_cuda_Relu_Convolution_56_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_cuda_Relu_Convolution_56<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, output0, output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1161_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2407_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3088_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1162_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2410_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3090_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1163_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2413_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3092_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_1157_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Slice_1159_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1182_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1183_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1184_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_1160_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1165_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087<<<dim3(1,
// 16, 4), dim3(8, 1, 16), 0, 0>>>(DepthwiseConv2dNative_1161_0,
// Constant_2407_0, Constant_3088_0, Relu_1182_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3089<<<dim3(1,
// 16, 4), dim3(8, 1, 16), 0, 0>>>(DepthwiseConv2dNative_1162_0,
// Constant_2410_0, Constant_3090_0, Relu_1183_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3091<<<dim3(1,
// 16, 4), dim3(8, 1, 16), 0, 0>>>(DepthwiseConv2dNative_1163_0,
// Constant_2413_0, Constant_3092_0, Relu_1184_0);
// Add_float_float_float_cuda_Add_1160<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(AvgPool_1157_0, AvgPool_1157_0, Add_1160_0);
// Relu_float_float_cuda_Relu_1165<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Slice_1159_0, Relu_1165_0); Deduped function map: <src_function_name :
// deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3089 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3091 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087

// Node name:	Matched_Pattern_3087
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1161_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2407_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3088_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1182_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 1, 16);
  const dim3 gridDim(1, 16, 4);
  const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[2];

      compute1[0] = 0.000000e+00f;
      compute1[1] = 0.000000e+00f;
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[(((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                  (((int)threadIdx.x) * 2))];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  16)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  17)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8192)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8193)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  32)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  33)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12288)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12289)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  48)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  49)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                (((int)blockIdx.y) * 16)) +
               (((int)threadIdx.x) * 2))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]),
              0.000000e+00f);
      compute[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.x) * 2)) +
               1)] =
          max((compute1[1] +
               input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
// Node name:	Add_1160
// Description:	Add
// Input:
//	- name: AvgPool_1157_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_1157_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1160_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1160_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Relu_1165
// Description:	Relu
// Input:
//	- name: Slice_1159_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1165_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_1165_block_kernel(float *input0, float *output0,
                                             int thread_id, int block_id,
                                             char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_Add_Relu_95(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {

  __shared__ char shared_buffer[2048];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_1160_block_kernel(
        input9, input9, output3, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 63) {
    Relu_float_float_cuda_Relu_1165_block_kernel(
        input10, output4, threadIdx.x, blockIdx.x - 32, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(
        input0, input1, input2, output0, threadIdx.x, blockIdx.x - 64,
        shared_buffer);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(
        input3, input4, input5, output1, threadIdx.x, blockIdx.x - 128,
        shared_buffer);
  } else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3087_block_kernel(
        input6, input7, input8, output2, threadIdx.x, blockIdx.x - 192,
        shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_Add_Relu_95_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_Add_Relu_95<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
