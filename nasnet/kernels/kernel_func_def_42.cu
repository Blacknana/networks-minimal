#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
// Node name:	Constant_2224
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2224_0	type: float	shape: Shape{64, 192, 1, 1}
void Constant_float_cuda_Constant_2224(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2224_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2224_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[49152];
  bin_file.read(tmp_mem, 49152);
  hipMemcpyAsync(output0, tmp_mem, 49152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2149
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2149_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2149(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2149_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2149_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2023
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2023_0	type: float	shape: Shape{32, 96, 1, 1}
void Constant_float_cuda_Constant_2023(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2023_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2023_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12288];
  bin_file.read(tmp_mem, 12288);
  hipMemcpyAsync(output0, tmp_mem, 12288, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2356
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2356_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2356(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2356_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2356_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_147
// Description:	Constant
// Input:
// Output:
//	- name: Constant_147_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_147(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_147_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_147_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2949
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2949_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2949(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2949_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2949_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2811
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2811_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2811(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2811_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2811_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2212
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2212_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2212(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2212_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2212_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3004
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3004_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3004(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3004_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3004_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_141
// Description:	Constant
// Input:
// Output:
//	- name: Constant_141_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_141(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_141_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_141_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[6400];
  bin_file.read(tmp_mem, 6400);
  hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3000
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3000_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3000(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3000_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3000_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2053
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2053_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2053(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2053_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2053_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2782
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2782_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2782(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2782_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2782_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: AvgPool_1674_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1606_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_264_0	type: float	shape: Shape{3, 3, 128, 1}
//	- name: Constant_186_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Relu_1700_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_414_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Relu_1698_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_45_0	type: float	shape: Shape{3, 3, 128, 1}
//	- name: Relu_1699_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_445_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: Add_1680_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1681_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1682_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1705_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1703_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1704_0	type: float	shape: Shape{1,
//128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_1680<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(AvgPool_1674_0, BatchNormInference_1606_0, Add_1680_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1675_0, Constant_264_0,
// DepthwiseConv2dNative_1681_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1682<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1675_0, Constant_186_0,
// DepthwiseConv2dNative_1682_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1705<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1700_0, Constant_414_0,
// DepthwiseConv2dNative_1705_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1703<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1698_0, Constant_45_0,
// DepthwiseConv2dNative_1703_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1704<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1699_0, Constant_445_0,
// DepthwiseConv2dNative_1704_0); Deduped function map: <src_function_name :
// deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1705 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1682
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1703 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1704 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681

// Node name:	Add_1680
// Description:	Add
// Input:
//	- name: AvgPool_1674_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1606_0	type: float	shape: Shape{1,
//128, 8, 8}
// Output:
//	- name: Add_1680_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1680_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	DepthwiseConv2dNative_1681
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_264_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1681_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_1682
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_186_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1682_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1682_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_170(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Add_float_float_float_cuda_Add_1680_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 79) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 16, NULL);
  } else if ((int)blockIdx.x >= 80 && (int)blockIdx.x <= 143) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1682_block_kernel(
        input2, input4, output2, threadIdx.x, blockIdx.x - 80, NULL);
  } else if ((int)blockIdx.x >= 144 && (int)blockIdx.x <= 207) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1682_block_kernel(
        input5, input6, output3, threadIdx.x, blockIdx.x - 144, NULL);
  } else if ((int)blockIdx.x >= 208 && (int)blockIdx.x <= 271) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681_block_kernel(
        input7, input8, output4, threadIdx.x, blockIdx.x - 208, NULL);
  } else if ((int)blockIdx.x >= 272 && (int)blockIdx.x <= 335) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1681_block_kernel(
        input9, input10, output5, threadIdx.x, blockIdx.x - 272, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_170_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_170<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4, output5);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1580_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2644_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: DepthwiseConv2dNative_1581_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2647_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1589_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Convolution_1591_0	type: float	shape: Shape{1, 128, 8,
//8}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_1589<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1580_0, Constant_2644_0,
// Convolution_1589_0);
// Convolution_float_float_float_cuda_Convolution_1591<<<dim3(1, 4, 16), dim3(8,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_1581_0, Constant_2647_0,
// Convolution_1591_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_1591 :
// Convolution_float_float_float_cuda_Convolution_1589

// Node name:	Convolution_1589
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_1580_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2644_0	type: float	shape: Shape{128, 128, 1, 1}
// Output:
//	- name: Convolution_1589_0	type: float	shape: Shape{1, 128, 8,
//8}
__device__ __forceinline__ void
Convolution_float_float_float_cuda_Convolution_1589_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute_local[1];

      compute_local[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] =
          (compute_local[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] = compute_local[0];
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_155(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_1589_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_1589_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64, shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_155_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_155<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
