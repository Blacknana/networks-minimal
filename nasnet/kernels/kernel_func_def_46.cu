#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_2317
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2317_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2317(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2317_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2317_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_414
// Description:	Constant
// Input:
// Output:
//	- name: Constant_414_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_414(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_414_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_414_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_58
// Description:	Constant
// Input:
// Output:
//	- name: Constant_58_0	type: float	shape: Shape{3, 3, 96, 1}
void Constant_float_cuda_Constant_58(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_58_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_58_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3456];
  bin_file.read(tmp_mem, 3456);
  hipMemcpyAsync(output0, tmp_mem, 3456, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2900
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2900_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2900(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2900_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2900_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_441
// Description:	Constant
// Input:
// Output:
//	- name: Constant_441_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_441(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_441_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_441_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2506
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2506_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2506(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2506_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2506_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3104
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3104_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3104(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3104_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3104_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_384
// Description:	Constant
// Input:
// Output:
//	- name: Constant_384_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_384(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_384_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_384_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2996
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2996_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2996(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2996_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2996_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2125
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2125_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2125(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2125_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2125_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2074
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2074_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2074(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2074_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2074_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: BatchNormInference_641_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Relu_644_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_62_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Constant_266_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_354_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: Slice_643_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_650_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_648_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_649_0	type: float	shape: Shape{1,
//32, 32, 32}
// Fused functions:
// Slice_float_float_cuda_Slice_643<<<dim3(512, 1, 1), dim3(64, 1, 1), 0,
// 0>>>(BatchNormInference_641_0, Slice_643_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_650<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_644_0, Constant_62_0,
// DepthwiseConv2dNative_650_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_648<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_644_0, Constant_266_0,
// DepthwiseConv2dNative_648_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_649<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_644_0, Constant_354_0,
// DepthwiseConv2dNative_649_0); Deduped function map: <src_function_name :
// deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_649 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_650

// Node name:	Slice_643
// Description:	Slice
// Input:
//	- name: BatchNormInference_641_0	type: float	shape: Shape{1,
//32, 32, 32}
// Output:
//	- name: Slice_643_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ void
Slice_float_float_cuda_Slice_643_block_kernel(float *input0, float *output0,
                                              int thread_id, int block_id,
                                              char *shared_buffer) {
  if (thread_id >= 64) {
    return;
  }
  const dim3 blockDim(64, 1, 1);
  const dim3 gridDim(512, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < 32768) {
    uint32_t input_strides[] = {32768, 1024, 32, 1};
    uint32_t output_strides[] = {32768, 1024, 32, 1};
    uint32_t lower_bounds[] = {0, 0, 0, 0};
    uint32_t slice_strides[] = {1, 1, 1, 1};
    uint32_t input_idx = 0;
    uint32_t output_idx = tid;
    input_idx += (((output_idx / output_strides[0]) * slice_strides[0]) +
                  lower_bounds[0]) *
                 input_strides[0];
    output_idx %= output_strides[0];
    input_idx += (((output_idx / output_strides[1]) * slice_strides[1]) +
                  lower_bounds[1]) *
                 input_strides[1];
    output_idx %= output_strides[1];
    input_idx += (((output_idx / output_strides[2]) * slice_strides[2]) +
                  lower_bounds[2]) *
                 input_strides[2];
    output_idx %= output_strides[2];
    input_idx += (((output_idx / output_strides[3]) * slice_strides[3]) +
                  lower_bounds[3]) *
                 input_strides[3];
    output0[tid] = input0[input_idx];
  }
}
__device__ __forceinline__ static void
fused2_DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_650_block_kernel(
    float *input0, float *input1, float *input2, float *output0, float *output1,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    S sum2 = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
          sum2 += static_cast<S>(__ldg(input + input_offset)) *
                  static_cast<S>(__ldg(input2 + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
            sum2 += static_cast<S>(__ldg(input + input_offset)) *
                    static_cast<S>(__ldg(input2 + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
    output1[thread_id] = static_cast<S>(sum2);
  }
}
// Node name:	DepthwiseConv2dNative_650
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_644_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_62_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_650_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_650_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_648
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_644_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_266_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_648_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_648_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_20(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *output0, float *output1, float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 255) {
    fused2_DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_650_block_kernel(
        input1, input2, input4, output1, output3, threadIdx.x, blockIdx.x - 0,
        NULL);
  } else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 511) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_648_block_kernel(
        input1, input3, output2, threadIdx.x, blockIdx.x - 256, NULL);
  }
  // else if((int)blockIdx.x >= 512 && (int)blockIdx.x <= 767)
  //{
  //    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_650_block_kernel(input1,
  //    input4, output3, threadIdx.x, blockIdx.x - 512, NULL);
  //}
  else if ((int)blockIdx.x >= 768 - 256 && (int)blockIdx.x <= 1279 - 256) {
    Slice_float_float_cuda_Slice_643_block_kernel(input0, output0, threadIdx.x,
                                                  blockIdx.x - 768 + 256, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_20_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *output0, float *output1, float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_cuda_Slice_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_20<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_1671_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1673_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1678_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2698_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3174_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1676_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2692_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3170_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1677_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2695_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3172_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1679_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1700_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1698_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1699_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Relu_float_float_cuda_Relu_1675<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Slice_1671_0, Relu_1675_0);
// Add_float_float_float_cuda_Add_1679<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(AvgPool_1673_0, AvgPool_1673_0, Add_1679_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1678_0, Constant_2698_0,
// Constant_3174_0, Relu_1700_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3169<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1676_0, Constant_2692_0,
// Constant_3170_0, Relu_1698_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3171<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1677_0, Constant_2695_0,
// Constant_3172_0, Relu_1699_0); Deduped function map: <src_function_name :
// deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3169 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3171 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173

// Node name:	Relu_1675
// Description:	Relu
// Input:
//	- name: Slice_1671_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1675_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_1675_block_kernel(float *input0, float *output0,
                                             int thread_id, int block_id,
                                             char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Add_1679
// Description:	Add
// Input:
//	- name: AvgPool_1673_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1673_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1679_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1679_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Matched_Pattern_3173
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1678_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2698_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3174_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1700_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[1];

      compute1[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_169(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Relu_float_float_cuda_Relu_1675_block_kernel(input0, output0, threadIdx.x,
                                                 blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_1679_block_kernel(
        input1, input1, output1, threadIdx.x, blockIdx.x - 16, shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(
        input2, input3, input4, output2, threadIdx.x, blockIdx.x - 32,
        shared_buffer);
  } else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(
        input5, input6, input7, output3, threadIdx.x, blockIdx.x - 96,
        shared_buffer);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3173_block_kernel(
        input8, input9, input10, output4, threadIdx.x, blockIdx.x - 160,
        shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_169_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Add_Matched_Pattern_Matched_Pattern_Matched_Pattern_169<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Constant_2909_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1204_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Slice_1159_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_2788_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_1206_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: BatchNormInference_1143_0	type: float	shape: Shape{1,
//64, 16, 16}
// Output:
//	- name: Add_1210_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Add_1211_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// FusedKernel_float_float_float_float_cuda_Add_Add_42<<<dim3(32, 1, 1),
// dim3(512, 1, 1), 0, 0>>>(Convolution_1204_0, Constant_2909_0, Slice_1159_0,
// Add_1210_0); FusedKernel_float_float_float_float_cuda_Add_Add_43<<<dim3(32,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1206_0, Constant_2788_0,
// BatchNormInference_1143_0, Add_1211_0); Deduped function map:
// <src_function_name : deduped_function_name>
// FusedKernel_float_float_float_float_cuda_Add_Add_43 :
// FusedKernel_float_float_float_float_cuda_Add_Add_42

// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1204_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Constant_2909_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Slice_1159_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_1210_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Add_float_float_float_cuda_Add_2433<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1204_0, Constant_2909_0, BatchNormInference_1208_0);
// Add_float_float_float_cuda_Add_1210<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1208_0, Slice_1159_0, Add_1210_0);
__device__ __forceinline__ void
FusedKernel_float_float_float_float_cuda_Add_Add_42_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(temp0, input2[tid]);
  output0[tid] = temp1;
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_100(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *output0, float *output1) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31) {
    FusedKernel_float_float_float_float_cuda_Add_Add_42_block_kernel(
        input1, input0, input2, output0, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 63) {
    FusedKernel_float_float_float_float_cuda_Add_Add_42_block_kernel(
        input4, input3, input5, output1, threadIdx.x, blockIdx.x - 32, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_100_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *output0, float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_cuda_fused_kernel_fused_kernel_100<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, output0, output1);
}
// Node name:	Constant_2266
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2266_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2266(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2266_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2266_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
