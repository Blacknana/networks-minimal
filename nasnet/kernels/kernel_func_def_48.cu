#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
// Node name:	Constant_2830
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2830_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2830(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2830_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2830_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3146
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3146_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3146(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3146_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3146_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2059
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2059_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2059(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2059_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2059_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_179
// Description:	Constant
// Input:
// Output:
//	- name: Constant_179_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_179(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_179_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_179_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2868
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2868_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2868(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2868_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2868_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2925
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2925_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2925(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2925_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2925_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_309
// Description:	Constant
// Input:
// Output:
//	- name: Constant_309_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_309(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_309_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_309_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_89
// Description:	Constant
// Input:
// Output:
//	- name: Constant_89_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_89(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_89_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_89_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2837
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2837_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2837(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2837_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2837_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3022
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3022_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_3022(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3022_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3022_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3078
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3078_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3078(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3078_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3078_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1097_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_368_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Constant_321_0	type: float	shape: Shape{3, 3, 64, 1}
//	- name: AvgPool_1098_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: BatchNormInference_1029_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Relu_1123_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_200_0	type: float	shape: Shape{3, 3, 64, 1}
//	- name: Relu_1121_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_230_0	type: float	shape: Shape{5, 5, 64, 1}
//	- name: Relu_1122_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_78_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1103_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: DepthwiseConv2dNative_1104_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Add_1105_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1128_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: DepthwiseConv2dNative_1126_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: DepthwiseConv2dNative_1127_0	type: float	shape: Shape{1,
//64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1103<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1097_0, Constant_368_0,
// DepthwiseConv2dNative_1103_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1097_0, Constant_321_0,
// DepthwiseConv2dNative_1104_0); Add_float_float_float_cuda_Add_1105<<<dim3(32,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1098_0, BatchNormInference_1029_0,
// Add_1105_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1128<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1123_0, Constant_200_0,
// DepthwiseConv2dNative_1128_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1126<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1121_0, Constant_230_0,
// DepthwiseConv2dNative_1126_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1127<<<dim3(128,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1122_0, Constant_78_0,
// DepthwiseConv2dNative_1127_0); Deduped function map: <src_function_name :
// deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1128 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1126 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1103
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1127 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104

// Node name:	DepthwiseConv2dNative_1103
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1097_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_368_0	type: float	shape: Shape{5, 5, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1103_0	type: float	shape: Shape{1,
//64, 16, 16}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1103_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 64;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 16;
  const int out_width = 16;
  const int out_depth = 64;
  const int num_outputs = 16384;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_1104
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1097_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Constant_321_0	type: float	shape: Shape{3, 3, 64, 1}
// Output:
//	- name: DepthwiseConv2dNative_1104_0	type: float	shape: Shape{1,
//64, 16, 16}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(128, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 16;
  const int in_width = 16;
  const int in_depth = 64;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 16;
  const int out_width = 16;
  const int out_depth = 64;
  const int num_outputs = 16384;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	Add_1105
// Description:	Add
// Input:
//	- name: AvgPool_1098_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: BatchNormInference_1029_0	type: float	shape: Shape{1,
//64, 16, 16}
// Output:
//	- name: Add_1105_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1105_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_87(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_1105_block_kernel(
        input3, input4, output2, threadIdx.x, blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 159) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1103_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 32, NULL);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 287) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104_block_kernel(
        input0, input2, output1, threadIdx.x, blockIdx.x - 160, NULL);
  } else if ((int)blockIdx.x >= 288 && (int)blockIdx.x <= 415) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104_block_kernel(
        input5, input6, output3, threadIdx.x, blockIdx.x - 288, NULL);
  } else if ((int)blockIdx.x >= 416 && (int)blockIdx.x <= 543) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1103_block_kernel(
        input7, input8, output4, threadIdx.x, blockIdx.x - 416, NULL);
  } else if ((int)blockIdx.x >= 544 && (int)blockIdx.x <= 671) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1104_block_kernel(
        input9, input10, output5, threadIdx.x, blockIdx.x - 544, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_87_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4, float *output5) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_Add_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_87<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4, output5);
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1308_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_221_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Relu_1306_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_169_0	type: float	shape: Shape{7, 7, 128, 1}
//	- name: Relu_1307_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_258_0	type: float	shape: Shape{7, 7, 128, 1}
//	- name: Slice_1284_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: DepthwiseConv2dNative_1313_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1311_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1312_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Relu_1288_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1313<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1308_0, Constant_221_0,
// DepthwiseConv2dNative_1313_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1311<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1306_0, Constant_169_0,
// DepthwiseConv2dNative_1311_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1312<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1307_0, Constant_258_0,
// DepthwiseConv2dNative_1312_0); Relu_float_float_cuda_Relu_1288<<<dim3(32, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Slice_1284_0, Relu_1288_0); Deduped function
// map: <src_function_name : deduped_function_name>
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1312 :
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1311

// Node name:	DepthwiseConv2dNative_1313
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1308_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_221_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1313_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1313_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_1311
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1306_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_169_0	type: float	shape: Shape{7, 7, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1311_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1311_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 7;
  const int filter_width = 7;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 3;
  const int pad_width = 3;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	Relu_1288
// Description:	Relu
// Input:
//	- name: Slice_1284_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1288_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_1288_block_kernel(float *input0, float *output0,
                                             int thread_id, int block_id,
                                             char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Relu_113(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *output0, float *output1,
    float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31) {
    Relu_float_float_cuda_Relu_1288_block_kernel(input6, output3, threadIdx.x,
                                                 blockIdx.x - 0, NULL);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1313_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 32, NULL);
  } else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1311_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 96, NULL);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1311_block_kernel(
        input4, input5, output2, threadIdx.x, blockIdx.x - 160, NULL);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Relu_113_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *output0, float *output1,
    float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_DepthwiseConv2dNative_Relu_113<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, output0, output1, output2,
                                    output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1423_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2548_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3130_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1425_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2554_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3134_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1424_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2551_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3132_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1419_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Slice_1421_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1444_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1446_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1445_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1422_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1426_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1423_0, Constant_2548_0,
// Constant_3130_0, Relu_1444_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3133<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1425_0, Constant_2554_0,
// Constant_3134_0, Relu_1446_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3131<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1424_0, Constant_2551_0,
// Constant_3132_0, Relu_1445_0); Add_float_float_float_cuda_Add_1422<<<dim3(16,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_1419_0, AvgPool_1419_0, Add_1422_0);
// Relu_float_float_cuda_Relu_1426<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Slice_1421_0, Relu_1426_0); Deduped function map: <src_function_name :
// deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3133 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3131 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129

// Node name:	Matched_Pattern_3129
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1423_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2548_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3130_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1444_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[1];

      compute1[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
// Node name:	Add_1422
// Description:	Add
// Input:
//	- name: AvgPool_1419_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: AvgPool_1419_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1422_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Add_float_float_float_cuda_Add_1422_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Relu_1426
// Description:	Relu
// Input:
//	- name: Slice_1421_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1426_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ void
Relu_float_float_cuda_Relu_1426_block_kernel(float *input0, float *output0,
                                             int thread_id, int block_id,
                                             char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}

extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_Add_Relu_133(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {

  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Add_float_float_float_cuda_Add_1422_block_kernel(
        input9, input9, output3, threadIdx.x, blockIdx.x - 0, shared_buffer);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Relu_float_float_cuda_Relu_1426_block_kernel(
        input10, output4, threadIdx.x, blockIdx.x - 16, shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129_block_kernel(
        input0, input1, input2, output0, threadIdx.x, blockIdx.x - 32,
        shared_buffer);
  } else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129_block_kernel(
        input3, input4, input5, output1, threadIdx.x, blockIdx.x - 96,
        shared_buffer);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3129_block_kernel(
        input6, input7, input8, output2, threadIdx.x, blockIdx.x - 160,
        shared_buffer);
  }
}

extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_Add_Relu_133_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_Add_Relu_133<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	Dot_1760
// Description:	Dot
// Input:
//	- name: Divide_1759_0	type: float	shape: Shape{1, 768}
//	- name: Constant_422_0	type: float	shape: Shape{768, 10}
// Output:
//	- name: Dot_1760_0	type: float	shape: Shape{1, 10}
void Dot_float_float_float_cuda_lib_Dot_1760(hipblasHandle_t cublas_handle,
                                             float *input0, float *input1,
                                             float *output0) {
  const float alpha = 1.0;
  const float beta = 0;
  CUBLAS_SAFE_CALL(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 10, 1,
                               768, &alpha, static_cast<const float *>(input1),
                               10, static_cast<const float *>(input0), 768,
                               &beta, static_cast<float *>(output0), 10));
}
