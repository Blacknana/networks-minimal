#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_3178
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3178_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3178(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3178_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3178_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2892
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2892_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_2892(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2892_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2892_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_437
// Description:	Constant
// Input:
// Output:
//	- name: Constant_437_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_437(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_437_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_437_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_186
// Description:	Constant
// Input:
// Output:
//	- name: Constant_186_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_186(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_186_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_186_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3144
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3144_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3144(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3144_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3144_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2853
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2853_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2853(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2853_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2853_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_425
// Description:	Constant
// Input:
// Output:
//	- name: Constant_425_0	type: float	shape: Shape{5, 5, 64, 1}
void Constant_float_cuda_Constant_425(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_425_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_425_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[6400];
  bin_file.read(tmp_mem, 6400);
  hipMemcpyAsync(output0, tmp_mem, 6400, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2988
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2988_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2988(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2988_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2988_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2137
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2137_0	type: float	shape: Shape{32, 32, 1, 1}
void Constant_float_cuda_Constant_2137(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2137_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2137_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4096];
  bin_file.read(tmp_mem, 4096);
  hipMemcpyAsync(output0, tmp_mem, 4096, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2230
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2230_0	type: float	shape: Shape{64, 64, 1, 1}
void Constant_float_cuda_Constant_2230(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2230_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2230_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[16384];
  bin_file.read(tmp_mem, 16384);
  hipMemcpyAsync(output0, tmp_mem, 16384, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_799_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_237_0	type: float	shape: Shape{3, 3, 32, 1}
//	- name: Relu_800_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_431_0	type: float	shape: Shape{5, 5, 32, 1}
//	- name: Constant_2844_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_807_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Convolution_811_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2771_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_809_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2783_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: DepthwiseConv2dNative_804_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: DepthwiseConv2dNative_805_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Add_821_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: BatchNormInference_817_0	type: float	shape: Shape{1,
//32, 32, 32}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_804<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_799_0, Constant_237_0,
// DepthwiseConv2dNative_804_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_805<<<dim3(256,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_800_0, Constant_431_0,
// DepthwiseConv2dNative_805_0);
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_20<<<dim3(64, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_807_0, Constant_2844_0,
// Convolution_811_0, Constant_2771_0, Add_821_0);
// Add_float_float_float_cuda_Add_2214<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_809_0, Constant_2783_0, BatchNormInference_817_0); Deduped
// function map: <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_804
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_799_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_237_0	type: float	shape: Shape{3, 3, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_804_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ static void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_804_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_805
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_800_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Constant_431_0	type: float	shape: Shape{5, 5, 32, 1}
// Output:
//	- name: DepthwiseConv2dNative_805_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ static void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_805_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(256, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 32;
  const int in_width = 32;
  const int in_depth = 32;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 32;
  const int out_width = 32;
  const int out_depth = 32;
  const int num_outputs = 32768;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_807_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2844_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Convolution_811_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2771_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_821_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2211<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_807_0, Constant_2844_0, BatchNormInference_816_0);
// Add_float_float_float_cuda_Add_2217<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_811_0, Constant_2771_0, BatchNormInference_818_0);
// Add_float_float_float_cuda_Add_821<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_816_0, BatchNormInference_818_0, Add_821_0);
__device__ __forceinline__ static void
FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_20_block_kernel(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(input2[tid], input3[tid]);
  float temp2 = add(temp0, temp1);
  output0[tid] = temp2;
}
// Node name:	Add_2214
// Description:	Add
// Input:
//	- name: Convolution_809_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Constant_2783_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: BatchNormInference_817_0	type: float	shape: Shape{1,
//32, 32, 32}
__device__ __forceinline__ static void
Add_float_float_float_cuda_Add_2214_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_fused_kernel_Add_42(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 255) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_804_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
  } else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 511) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_805_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 256 + 0, NULL);
  } else if ((int)blockIdx.x >= 512 && (int)blockIdx.x <= 575) {
    FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_20_block_kernel(
        input5, input4, input6, input7, output2, threadIdx.x,
        blockIdx.x - 512 + 0, NULL);
  } else if ((int)blockIdx.x >= 576 && (int)blockIdx.x <= 639) {
    Add_float_float_float_cuda_Add_2214_block_kernel(
        input8, input9, output3, threadIdx.x, blockIdx.x - 576 + 0, NULL);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_fused_kernel_Add_42_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_fused_kernel_Add_42<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, input7, input8, input9,
                                    output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: Convolution_1321_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2924_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1317_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2923_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1319_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2922_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1296_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2488_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3116_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_1297_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2491_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3118_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: BatchNormInference_1326_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: BatchNormInference_1324_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: BatchNormInference_1325_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Relu_1314_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_1315_0	type: float	shape: Shape{1, 64, 16, 16}
// Fused functions:
// Add_float_float_float_cuda_Add_2502<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1321_0, Constant_2924_0, BatchNormInference_1326_0);
// Add_float_float_float_cuda_Add_2496<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1317_0, Constant_2923_0, BatchNormInference_1324_0);
// Add_float_float_float_cuda_Add_2499<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1319_0, Constant_2922_0, BatchNormInference_1325_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3115<<<dim3(1,
// 16, 4), dim3(8, 1, 16), 0, 0>>>(DepthwiseConv2dNative_1296_0,
// Constant_2488_0, Constant_3116_0, Relu_1314_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3117<<<dim3(1,
// 16, 4), dim3(8, 1, 16), 0, 0>>>(DepthwiseConv2dNative_1297_0,
// Constant_2491_0, Constant_3118_0, Relu_1315_0); Deduped function map:
// <src_function_name : deduped_function_name>
// Add_float_float_float_cuda_Add_2496 : Add_float_float_float_cuda_Add_2502
// Add_float_float_float_cuda_Add_2499 : Add_float_float_float_cuda_Add_2502
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3117 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3115

// Node name:	Add_2502
// Description:	Add
// Input:
//	- name: Convolution_1321_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2924_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1326_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ static void
Add_float_float_float_cuda_Add_2502_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Matched_Pattern_3115
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1296_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2488_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3116_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_1314_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ static void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3115_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 1, 16);
  const dim3 gridDim(1, 16, 4);
  const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[2];

      compute1[0] = 0.000000e+00f;
      compute1[1] = 0.000000e+00f;
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[(((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                  (((int)threadIdx.x) * 2))];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  16)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  17)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8192)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8193)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  32)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  33)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12288)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12289)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  48)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  49)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                (((int)blockIdx.y) * 16)) +
               (((int)threadIdx.x) * 2))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]),
              0.000000e+00f);
      compute[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.x) * 2)) +
               1)] =
          max((compute1[1] +
               input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Add_Add_Matched_Pattern_Matched_Pattern_115(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *input11, float *output0, float *output1,
    float *output2, float *output3, float *output4) {
  __shared__ char shared_buffer[2048];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 15) {
    Add_float_float_float_cuda_Add_2502_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 16 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_2502_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 16 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 47) {
    Add_float_float_float_cuda_Add_2502_block_kernel(
        input4, input5, output2, threadIdx.x, blockIdx.x - 32 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 48 && (int)blockIdx.x <= 111) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3115_block_kernel(
        input6, input7, input8, output3, threadIdx.x, blockIdx.x - 48 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 112 && (int)blockIdx.x <= 175) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3115_block_kernel(
        input9, input10, input11, output4, threadIdx.x, blockIdx.x - 112 + 0,
        shared_buffer);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Add_Add_Matched_Pattern_Matched_Pattern_115_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *input11, float *output0, float *output1,
    float *output2, float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Add_Add_Matched_Pattern_Matched_Pattern_115<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, input11, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: Add_903_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_937_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_946_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2278_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3056_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: DepthwiseConv2dNative_934_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2281_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_935_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2284_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: DepthwiseConv2dNative_936_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2287_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Add_945_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Relu_954_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: Convolution_940_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Convolution_942_0	type: float	shape: Shape{1, 64, 16,
//16}
//	- name: Convolution_944_0	type: float	shape: Shape{1, 64, 16,
//16}
// Fused functions:
// Add_float_float_float_cuda_Add_945<<<dim3(32, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Add_903_0, AvgPool_937_0, Add_945_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3055<<<dim3(1,
// 16, 4), dim3(8, 1, 16), 0, 0>>>(DepthwiseConv2dNative_946_0, Constant_2278_0,
// Constant_3056_0, Relu_954_0);
// Convolution_float_float_float_cuda_Convolution_940<<<dim3(1, 16, 4), dim3(8,
// 1, 16), 0, 0>>>(DepthwiseConv2dNative_934_0, Constant_2281_0,
// Convolution_940_0);
// Convolution_float_float_float_cuda_Convolution_942<<<dim3(1, 16, 4), dim3(8,
// 1, 16), 0, 0>>>(DepthwiseConv2dNative_935_0, Constant_2284_0,
// Convolution_942_0);
// Convolution_float_float_float_cuda_Convolution_944<<<dim3(1, 16, 4), dim3(8,
// 1, 16), 0, 0>>>(DepthwiseConv2dNative_936_0, Constant_2287_0,
// Convolution_944_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_942 :
// Convolution_float_float_float_cuda_Convolution_940
// Convolution_float_float_float_cuda_Convolution_944 :
// Convolution_float_float_float_cuda_Convolution_940

// Node name:	Add_945
// Description:	Add
// Input:
//	- name: Add_903_0	type: float	shape: Shape{1, 64, 16, 16}
//	- name: AvgPool_937_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Add_945_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ static void
Add_float_float_float_cuda_Add_945_block_kernel(float *input0, float *input1,
                                                float *output0, int thread_id,
                                                int block_id,
                                                char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(32, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Matched_Pattern_3055
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_946_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2278_0	type: float	shape: Shape{64, 64, 1, 1}
//	- name: Constant_3056_0	type: float	shape: Shape{1, 64, 16, 16}
// Output:
//	- name: Relu_954_0	type: float	shape: Shape{1, 64, 16, 16}
__device__ __forceinline__ static void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3055_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 1, 16);
  const dim3 gridDim(1, 16, 4);
  const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[2];

      compute1[0] = 0.000000e+00f;
      compute1[1] = 0.000000e+00f;
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[(((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                  (((int)threadIdx.x) * 2))];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  16)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  17)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8192)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8193)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  32)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  33)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12288)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12289)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  48)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  49)];
      __syncthreads();
      compute1[0] = (compute1[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                                    input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                          input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute1[1] =
          (compute1[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                          input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                (((int)blockIdx.y) * 16)) +
               (((int)threadIdx.x) * 2))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]),
              0.000000e+00f);
      compute[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.x) * 2)) +
               1)] =
          max((compute1[1] +
               input2[((((int)blockIdx.z) * 16) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
// Node name:	Convolution_940
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_934_0	type: float	shape: Shape{1,
//64, 16, 16}
//	- name: Constant_2281_0	type: float	shape: Shape{64, 64, 1, 1}
// Output:
//	- name: Convolution_940_0	type: float	shape: Shape{1, 64, 16,
//16}
__device__ __forceinline__ static void
Convolution_float_float_float_cuda_Convolution_940_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 1, 16);
  const dim3 gridDim(1, 16, 4);
  const dim3 threadIdx(thread_id % 8, 0, thread_id / 8);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 16, block_id / 16);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute_local[2];

      compute_local[0] = 0.000000e+00f;
      compute_local[1] = 0.000000e+00f;
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[(((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[(((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                  (((int)threadIdx.x) * 2))];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  16)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  17)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8192)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  8193)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  32)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  33)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12288)];
      pad_temp_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                       1)] =
          input0[((((((int)threadIdx.z) * 256) + (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  12289)];
      input1_shared[((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  48)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.x) * 2)) +
                     1)] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 64)) +
                   (((int)threadIdx.x) * 2)) +
                  49)];
      __syncthreads();
      compute_local[0] =
          (compute_local[0] + (pad_temp_shared[(((int)threadIdx.x) * 2)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[1] =
          (compute_local[1] + (pad_temp_shared[((((int)threadIdx.x) * 2) + 1)] *
                               input1_shared[(((int)threadIdx.z) * 16)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 16)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 17)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 32)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 33)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 48)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 49)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 64)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 65)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 80)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 81)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 96)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 97)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 112)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 113)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 128)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 129)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 144)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 145)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 160)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 161)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 176)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 177)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 192)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 193)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 208)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 209)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 224)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 225)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute_local[0] = (compute_local[0] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 240)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute_local[1] = (compute_local[1] +
                          (pad_temp_shared[((((int)threadIdx.x) * 2) + 241)] *
                           input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                (((int)blockIdx.y) * 16)) +
               (((int)threadIdx.x) * 2))] = compute_local[0];
      compute[(((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.x) * 2)) +
               1)] = compute_local[1];
    }
  }
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Matched_Pattern_Convolution_Convolution_Convolution_63(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  __shared__ char shared_buffer[2048];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 31) {
    Add_float_float_float_cuda_Add_945_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 32 && (int)blockIdx.x <= 95) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3055_block_kernel(
        input2, input3, input4, output1, threadIdx.x, blockIdx.x - 32 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 96 && (int)blockIdx.x <= 159) {
    Convolution_float_float_float_cuda_Convolution_940_block_kernel(
        input5, input6, output2, threadIdx.x, blockIdx.x - 96 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 160 && (int)blockIdx.x <= 223) {
    Convolution_float_float_float_cuda_Convolution_940_block_kernel(
        input7, input8, output3, threadIdx.x, blockIdx.x - 160 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 224 && (int)blockIdx.x <= 287) {
    Convolution_float_float_float_cuda_Convolution_940_block_kernel(
        input9, input10, output4, threadIdx.x, blockIdx.x - 224 + 0,
        shared_buffer);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Matched_Pattern_Convolution_Convolution_Convolution_63_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Add_Matched_Pattern_Convolution_Convolution_Convolution_63<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_678_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2146_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: DepthwiseConv2dNative_679_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2149_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_687_0	type: float	shape: Shape{1, 32, 32,
//32}
//	- name: Convolution_689_0	type: float	shape: Shape{1, 32, 32,
//32}
// Fused functions:
// Convolution_float_float_float_cuda_Convolution_687<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_678_0, Constant_2146_0,
// Convolution_687_0);
// Convolution_float_float_float_cuda_Convolution_689<<<dim3(2, 16, 2), dim3(16,
// 2, 8), 0, 0>>>(DepthwiseConv2dNative_679_0, Constant_2149_0,
// Convolution_689_0); Deduped function map: <src_function_name :
// deduped_function_name> Convolution_float_float_float_cuda_Convolution_689 :
// Convolution_float_float_float_cuda_Convolution_687

// Node name:	Convolution_687
// Description:	Convolution
// Input:
//	- name: DepthwiseConv2dNative_678_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2146_0	type: float	shape: Shape{32, 32, 1, 1}
// Output:
//	- name: Convolution_687_0	type: float	shape: Shape{1, 32, 32,
//32}
__device__ __forceinline__ static void
Convolution_float_float_float_cuda_Convolution_687_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 2, 8);
  const dim3 gridDim(2, 16, 2);
  const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute_local[2];

      for (int ff_c_init = 0; ff_c_init < 2; ++ff_c_init) {
        compute_local[ff_c_init] = 0.000000e+00f;
      }
      for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) +
               (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                      (((int)threadIdx.y) * 1024)) +
                     (((int)blockIdx.y) * 64)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      4) *
                     32)) +
                   (((int)blockIdx.x) * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   15))];
        }
        input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       ((int)threadIdx.x))] =
            input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                      (((int)threadIdx.y) * 32)) +
                     (rc_outer * 16)) +
                    ((int)threadIdx.x))];
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff_c = 0; ff_c < 2; ++ff_c) {
            compute_local[ff_c] =
                (compute_local[ff_c] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.y) * 16)) +
                      ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 32) + (ff_c * 16)) +
                                 rc_inner)]));
          }
        }
      }
      for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 2;
           ++ff_inner_inner_inner) {
        compute[(
            ((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) +
                (ff_inner_inner_inner * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (((int)threadIdx.y) * 32)) +
             (((int)blockIdx.x) * 16)) +
            ((int)threadIdx.x))] = compute_local[ff_inner_inner_inner];
      }
    }
  }
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_25(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  __shared__ char shared_buffer[3072];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Convolution_float_float_float_cuda_Convolution_687_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Convolution_float_float_float_cuda_Convolution_687_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0,
        shared_buffer);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_25_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *output0,
    float *output1) {
  BlockFusionKernel_float_float_float_float_float_float_cuda_Convolution_Convolution_25<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, output0,
                                    output1);
}
// Node name:	 BlockFusion
// Input:
//	- name: Slice_769_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_775_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2194_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3026_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_777_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2200_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3030_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: AvgPool_770_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: DepthwiseConv2dNative_776_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2197_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3028_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_772_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_796_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_798_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Add_774_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: Relu_797_0	type: float	shape: Shape{1, 32, 32, 32}
// Fused functions:
// Relu_float_float_cuda_Relu_772<<<dim3(64, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Slice_769_0, Relu_772_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025<<<dim3(2,
// 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_775_0, Constant_2194_0,
// Constant_3026_0, Relu_796_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3029<<<dim3(2,
// 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_777_0, Constant_2200_0,
// Constant_3030_0, Relu_798_0); Add_float_float_float_cuda_Add_774<<<dim3(64,
// 1, 1), dim3(512, 1, 1), 0, 0>>>(AvgPool_770_0, AvgPool_770_0, Add_774_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3027<<<dim3(2,
// 16, 2), dim3(16, 2, 8), 0, 0>>>(DepthwiseConv2dNative_776_0, Constant_2197_0,
// Constant_3028_0, Relu_797_0); Deduped function map: <src_function_name :
// deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3029 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3027 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025

// Node name:	Relu_772
// Description:	Relu
// Input:
//	- name: Slice_769_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_772_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ static void
Relu_float_float_cuda_Relu_772_block_kernel(float *input0, float *output0,
                                            int thread_id, int block_id,
                                            char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      relu(input0[blockIdx.x * 512 + threadIdx.x]);
}
// Node name:	Matched_Pattern_3025
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_775_0	type: float	shape: Shape{1,
//32, 32, 32}
//	- name: Constant_2194_0	type: float	shape: Shape{32, 32, 1, 1}
//	- name: Constant_3026_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Relu_796_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ static void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 256) {
    return;
  }
  const dim3 blockDim(16, 2, 8);
  const dim3 gridDim(2, 16, 2);
  const dim3 threadIdx(thread_id % 16, thread_id / 16 % 2, thread_id / 32);
  const dim3 blockIdx(block_id % 2, block_id / 2 % 16, block_id / 32);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 2048);
  {
    float *compute = output0;
    {
      float compute1[2];

      for (int ff_init = 0; ff_init < 2; ++ff_init) {
        compute1[ff_init] = 0.000000e+00f;
      }
      for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
        __syncthreads();
        for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0;
             ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2;
             ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
          pad_temp_shared[(
              (((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 32)) +
               (((int)threadIdx.x) * 2)) +
              ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] =
              input0[(
                  ((((((rc_outer * 16384) + (((int)threadIdx.z) * 2048)) +
                      (((int)threadIdx.y) * 1024)) +
                     (((int)blockIdx.y) * 64)) +
                    ((((((int)threadIdx.x) * 2) +
                       ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >>
                      4) *
                     32)) +
                   (((int)blockIdx.x) * 16)) +
                  (((((int)threadIdx.x) * 2) +
                    ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) &
                   15))];
        }
        input1_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       ((int)threadIdx.x))] =
            input1[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                      (((int)threadIdx.y) * 32)) +
                     (rc_outer * 16)) +
                    ((int)threadIdx.x))];
        __syncthreads();
        for (int rc_inner = 0; rc_inner < 16; ++rc_inner) {
          for (int ff = 0; ff < 2; ++ff) {
            compute1[ff] =
                (compute1[ff] +
                 (pad_temp_shared[(
                      ((rc_inner * 32) + (((int)threadIdx.y) * 16)) +
                      ((int)threadIdx.x))] *
                  input1_shared[(((((int)threadIdx.z) * 32) + (ff * 16)) +
                                 rc_inner)]));
          }
        }
      }
      for (int i1_inner_inner_inner = 0; i1_inner_inner_inner < 2;
           ++i1_inner_inner_inner) {
        compute[(
            ((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 2048)) +
                (i1_inner_inner_inner * 1024)) +
               (((int)blockIdx.y) * 64)) +
              (((int)threadIdx.y) * 32)) +
             (((int)blockIdx.x) * 16)) +
            ((int)threadIdx.x))] =
            max((compute1[i1_inner_inner_inner] +
                 input2[(((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) +
                         i1_inner_inner_inner)]),
                0.000000e+00f);
      }
    }
  }
}
// Node name:	Add_774
// Description:	Add
// Input:
//	- name: AvgPool_770_0	type: float	shape: Shape{1, 32, 32, 32}
//	- name: AvgPool_770_0	type: float	shape: Shape{1, 32, 32, 32}
// Output:
//	- name: Add_774_0	type: float	shape: Shape{1, 32, 32, 32}
__device__ __forceinline__ static void
Add_float_float_float_cuda_Add_774_block_kernel(float *input0, float *input1,
                                                float *output0, int thread_id,
                                                int block_id,
                                                char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_39(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  __shared__ char shared_buffer[3072];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Relu_float_float_cuda_Relu_772_block_kernel(
        input0, output0, threadIdx.x, blockIdx.x - 0 + 0, shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025_block_kernel(
        input1, input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025_block_kernel(
        input4, input5, input6, output2, threadIdx.x, blockIdx.x - 128 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 255) {
    Add_float_float_float_cuda_Add_774_block_kernel(
        input7, input7, output3, threadIdx.x, blockIdx.x - 192 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 256 && (int)blockIdx.x <= 319) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3025_block_kernel(
        input8, input9, input10, output4, threadIdx.x, blockIdx.x - 256 + 0,
        shared_buffer);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_39_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *output0, float *output1, float *output2,
    float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Relu_Matched_Pattern_Matched_Pattern_Add_Matched_Pattern_39<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, output0, output1, output2, output3, output4);
}
