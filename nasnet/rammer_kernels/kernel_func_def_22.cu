#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT License.

#include "shared.h"
__device__ __forceinline__ float add(float x0, float x1) { return x0 + x1; }
__device__ __forceinline__ float relu(float x0) { return fmaxf(0, x0); }
// Node name:	Constant_29
// Description:	Constant
// Input:
// Output:
//	- name: Constant_29_0	type: float	shape: Shape{3, 3, 64, 1}
void Constant_float_cuda_Constant_29(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_29_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_29_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[2304];
  bin_file.read(tmp_mem, 2304);
  hipMemcpyAsync(output0, tmp_mem, 2304, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_235
// Description:	Constant
// Input:
// Output:
//	- name: Constant_235_0	type: float	shape: Shape{3, 3, 32, 1}
void Constant_float_cuda_Constant_235(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_235_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_235_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[1152];
  bin_file.read(tmp_mem, 1152);
  hipMemcpyAsync(output0, tmp_mem, 1152, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_450
// Description:	Constant
// Input:
// Output:
//	- name: Constant_450_0	type: float	shape: Shape{5, 5, 32, 1}
void Constant_float_cuda_Constant_450(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_450_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_450_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[3200];
  bin_file.read(tmp_mem, 3200);
  hipMemcpyAsync(output0, tmp_mem, 3200, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2980
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2980_0	type: float	shape: Shape{1, 32, 32, 32}
void Constant_float_cuda_Constant_2980(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2980_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2980_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[131072];
  bin_file.read(tmp_mem, 131072);
  hipMemcpyAsync(output0, tmp_mem, 131072, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_279
// Description:	Constant
// Input:
// Output:
//	- name: Constant_279_0	type: float	shape: Shape{5, 5, 128, 1}
void Constant_float_cuda_Constant_279(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_279_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_279_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12800];
  bin_file.read(tmp_mem, 12800);
  hipMemcpyAsync(output0, tmp_mem, 12800, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2936
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2936_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2936(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2936_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2936_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_376
// Description:	Constant
// Input:
// Output:
//	- name: Constant_376_0	type: float	shape: Shape{7, 7, 128, 1}
void Constant_float_cuda_Constant_376(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_376_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_376_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[25088];
  bin_file.read(tmp_mem, 25088);
  hipMemcpyAsync(output0, tmp_mem, 25088, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3114
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3114_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_3114(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3114_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3114_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_2960
// Description:	Constant
// Input:
// Output:
//	- name: Constant_2960_0	type: float	shape: Shape{1, 128, 8, 8}
void Constant_float_cuda_Constant_2960(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_2960_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_2960_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[32768];
  bin_file.read(tmp_mem, 32768);
  hipMemcpyAsync(output0, tmp_mem, 32768, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_3096
// Description:	Constant
// Input:
// Output:
//	- name: Constant_3096_0	type: float	shape: Shape{1, 64, 16, 16}
void Constant_float_cuda_Constant_3096(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_3096_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_3096_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[65536];
  bin_file.read(tmp_mem, 65536);
  hipMemcpyAsync(output0, tmp_mem, 65536, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_244
// Description:	Constant
// Input:
// Output:
//	- name: Constant_244_0	type: float	shape: Shape{3, 3, 128, 1}
void Constant_float_cuda_Constant_244(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_244_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_244_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[4608];
  bin_file.read(tmp_mem, 4608);
  hipMemcpyAsync(output0, tmp_mem, 4608, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	Constant_366
// Description:	Constant
// Input:
// Output:
//	- name: Constant_366_0	type: float	shape: Shape{7, 7, 64, 1}
void Constant_float_cuda_Constant_366(hipStream_t stream, float *output0) {
  std::ifstream bin_file("./Constant/Constant_366_0.bin",
                         std::ios::in | std::ios::binary);
  if (bin_file.fail()) {
    printf("Load Constant_366_0 failed.\n");
    exit(1);
  }
  char *tmp_mem = new char[12544];
  bin_file.read(tmp_mem, 12544);
  hipMemcpyAsync(output0, tmp_mem, 12544, hipMemcpyHostToDevice, stream);
  bin_file.close();
}
// Node name:	 BlockFusion
// Input:
//	- name: Relu_1575_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_402_0	type: float	shape: Shape{3, 3, 128, 1}
//	- name: Relu_1576_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_457_0	type: float	shape: Shape{5, 5, 128, 1}
//	- name: Constant_2949_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1583_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2951_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1587_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Convolution_1585_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2950_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: DepthwiseConv2dNative_1580_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: DepthwiseConv2dNative_1581_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Add_1597_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1593_0	type: float	shape: Shape{1,
//128, 8, 8}
// Fused functions:
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1580<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1575_0, Constant_402_0,
// DepthwiseConv2dNative_1580_0);
// DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1581<<<dim3(64,
// 1, 1), dim3(128, 1, 1), 0, 0>>>(Relu_1576_0, Constant_457_0,
// DepthwiseConv2dNative_1581_0);
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_62<<<dim3(16, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1583_0, Constant_2949_0,
// Convolution_1587_0, Constant_2951_0, Add_1597_0);
// Add_float_float_float_cuda_Add_2640<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1585_0, Constant_2950_0, BatchNormInference_1593_0); Deduped
// function map: <src_function_name : deduped_function_name>

// Node name:	DepthwiseConv2dNative_1580
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1575_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_402_0	type: float	shape: Shape{3, 3, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1580_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ static void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1580_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 3;
  const int filter_width = 3;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 1;
  const int pad_width = 1;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	DepthwiseConv2dNative_1581
// Description:	DepthwiseConv2dNative
// Input:
//	- name: Relu_1576_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_457_0	type: float	shape: Shape{5, 5, 128, 1}
// Output:
//	- name: DepthwiseConv2dNative_1581_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ static void
DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1581_block_kernel(
    float *input0, float *input1, float *output0, int thread_id, int block_id,
    char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(128, 1, 1);
  const dim3 gridDim(64, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);

  typedef float S;
  float *input = input0;
  float *filter = input1;
  float *output = output0;

  const int in_height = 8;
  const int in_width = 8;
  const int in_depth = 128;
  const int filter_height = 5;
  const int filter_width = 5;
  const int depth_multiplier = 1;
  const int stride = 1;
  const int pad_height = 2;
  const int pad_width = 2;
  const int out_height = 8;
  const int out_width = 8;
  const int out_depth = 128;
  const int num_outputs = 8192;

  for (uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
       thread_id < num_outputs; thread_id += blockDim.x * gridDim.x) {
    // Compute the indexes of this thread in the output.
    //
    // We want coalesced reads so we make sure that each warp reads
    // a contiguous chunk of memory.
    //
    // THIS IS PROBABLY WRONG, we are not doing coalesced reads
    // into the input, because of the depth multiplier division...
    const int out_col = thread_id % out_width;
    const int out_row = (thread_id / out_width) % out_height;
    const int out_channel = (thread_id / out_width / out_height) % out_depth;
    const int batch = thread_id / out_width / out_height / out_depth;

    // Compute the input depth and the index of depth multiplier
    // based off the output depth index that this thread is
    // computing n.
    const int in_channel = out_channel / depth_multiplier;
    const int multiplier = out_channel % depth_multiplier;

    // Data is stored in the following format (let's assume we
    // flatten the height and width into one contiguous dimension
    // called "P".
    //
    // B1C1P1 B1C1P2 ..... B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 ..... B2C2P1 B2C2P2 ....
    //
    // Each row contains in_depth * in_height * in_width values
    // for each sample in the batch.
    //
    // We can further flatten it into:
    //
    // B1C1P1 B1C1P2 .....
    // B1C2P1 B1C2P2 ....
    // B2C1P1 B2C1P2 .....
    // B2C2P1 B2C2P2 ....
    //
    // where each row is a contiguous array of all of the spatial
    // pixels for a given batch and input depth.  The following
    // loop #pragma unrolls across the filter dimensions for a given thread,
    // indexing into the filter value and the corresponding input
    // patch.
    //
    // We can compute the index into the patch once right here.
    const int input_offset_temp =
        (batch * in_depth + in_channel) * (in_height * in_width);

    // Finally, we can iterate over the spatial dimensions and perform the
    // convolution, writing into the output at the end.
    //
    // We perform an additional optimization, where we can determine
    // whether the patch fits within the image indices statically, and
    // avoid boundary checking within the loop.
    const int input_row_start = out_row * stride - pad_height;
    const int input_col_start = out_col * stride - pad_width;
    const int input_row_end = input_row_start + filter_height;
    const int input_col_end = input_col_start + filter_width;

    S sum = static_cast<S>(0);
    if (input_row_start >= 0 && input_col_start >= 0 &&
        input_row_end < in_height && input_col_end < in_width) {
// Loop that doesn't need to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;

          const int input_offset =
              (input_offset_temp) + (in_row * in_width) + in_col;
          const int filter_offset =
              multiplier +
              depth_multiplier *
                  (in_channel + in_depth * (filter_col + filter_offset_temp));
          sum += static_cast<S>(__ldg(input + input_offset)) *
                 static_cast<S>(__ldg(filter + filter_offset));
        }
      }
    } else {
// Loop that needs to check for boundary conditions.
#pragma unroll
      for (int filter_row = 0; filter_row < filter_height; ++filter_row) {
        const int in_row = input_row_start + filter_row;
        const int filter_offset_temp = filter_width * filter_row;
#pragma unroll
        for (int filter_col = 0; filter_col < filter_width; ++filter_col) {
          const int in_col = input_col_start + filter_col;
          // TODO(vrv): the in_row check can be done outside of this loop;
          // benchmark both methods to determine the better decision.
          if (in_row >= 0 && in_row < in_height && in_col >= 0 &&
              in_col < in_width) {
            const int in_col = input_col_start + filter_col;

            // input_offset_temp indexes into the start of memory
            // where the spatial data starts.
            const int input_offset =
                (input_offset_temp) + (in_row * in_width) + in_col;

            const int filter_offset =
                multiplier +
                depth_multiplier *
                    (in_channel + in_depth * (filter_col + filter_offset_temp));
            sum += static_cast<S>(__ldg(input + input_offset)) *
                   static_cast<S>(__ldg(filter + filter_offset));
          }
        }
      }
    }

    output[thread_id] = static_cast<S>(sum);
  }
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1583_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2949_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1587_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2951_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Add_1597_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2637<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1583_0, Constant_2949_0, BatchNormInference_1592_0);
// Add_float_float_float_cuda_Add_2643<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1587_0, Constant_2951_0, BatchNormInference_1594_0);
// Add_float_float_float_cuda_Add_1597<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1592_0, BatchNormInference_1594_0, Add_1597_0);
__device__ __forceinline__ static void
FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_62_block_kernel(
    float *input0, float *input1, float *input2, float *input3, float *output0,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(input2[tid], input3[tid]);
  float temp2 = add(temp0, temp1);
  output0[tid] = temp2;
}
// Node name:	Add_2640
// Description:	Add
// Input:
//	- name: Convolution_1585_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2950_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: BatchNormInference_1593_0	type: float	shape: Shape{1,
//128, 8, 8}
__device__ __forceinline__ static void
Add_float_float_float_cuda_Add_2640_block_kernel(float *input0, float *input1,
                                                 float *output0, int thread_id,
                                                 int block_id,
                                                 char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  output0[blockIdx.x * 512 + threadIdx.x] =
      add(input0[blockIdx.x * 512 + threadIdx.x],
          input1[blockIdx.x * 512 + threadIdx.x]);
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_fused_kernel_Add_154(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1580_block_kernel(
        input0, input1, output0, threadIdx.x, blockIdx.x - 0 + 0, NULL);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    DepthwiseConv2dNative_float_float_float_cuda_DepthwiseConv2dNative_1581_block_kernel(
        input2, input3, output1, threadIdx.x, blockIdx.x - 64 + 0, NULL);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 143) {
    FusedKernel_float_float_float_float_float_cuda_Add_Add_Add_62_block_kernel(
        input5, input4, input7, input6, output2, threadIdx.x,
        blockIdx.x - 128 + 0, NULL);
  } else if ((int)blockIdx.x >= 144 && (int)blockIdx.x <= 159) {
    Add_float_float_float_cuda_Add_2640_block_kernel(
        input8, input9, output3, threadIdx.x, blockIdx.x - 144 + 0, NULL);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_fused_kernel_Add_154_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *output0, float *output1, float *output2, float *output3) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_DepthwiseConv2dNative_DepthwiseConv2dNative_fused_kernel_Add_154<<<
      grids, blocks, mem, stream>>>(input0, input1, input2, input3, input4,
                                    input5, input6, input7, input8, input9,
                                    output0, output1, output2, output3);
}
// Node name:	 BlockFusion
// Input:
//	- name: DepthwiseConv2dNative_1363_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2518_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3122_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1364_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2521_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3124_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: DepthwiseConv2dNative_1365_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2524_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3126_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Constant_2868_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Convolution_1379_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: BatchNormInference_1324_0	type: float	shape: Shape{1,
//128, 8, 8}
// Output:
//	- name: Relu_1380_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1381_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1382_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1387_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Relu_1395_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1363_0, Constant_2518_0,
// Constant_3122_0, Relu_1380_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3123<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1364_0, Constant_2521_0,
// Constant_3124_0, Relu_1381_0);
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3125<<<dim3(1,
// 4, 16), dim3(8, 2, 8), 0, 0>>>(DepthwiseConv2dNative_1365_0, Constant_2524_0,
// Constant_3126_0, Relu_1382_0);
// FusedKernel_float_float_float_float_float_cuda_Add_Add_Relu_51<<<dim3(16, 1,
// 1), dim3(512, 1, 1), 0, 0>>>(Convolution_1379_0, Constant_2868_0,
// BatchNormInference_1324_0, Relu_1395_0, Add_1387_0); Deduped function map:
// <src_function_name : deduped_function_name>
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3123 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3125 :
// Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121

// Node name:	Matched_Pattern_3121
// Description:	Matched_Pattern
// Input:
//	- name: DepthwiseConv2dNative_1363_0	type: float	shape: Shape{1,
//128, 8, 8}
//	- name: Constant_2518_0	type: float	shape: Shape{128, 128, 1, 1}
//	- name: Constant_3122_0	type: float	shape: Shape{1, 128, 8, 8}
// Output:
//	- name: Relu_1380_0	type: float	shape: Shape{1, 128, 8, 8}
__device__ __forceinline__ static void
Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121_block_kernel(
    float *input0, float *input1, float *input2, float *output0, int thread_id,
    int block_id, char *shared_buffer) {
  if (thread_id >= 128) {
    return;
  }
  const dim3 blockDim(8, 2, 8);
  const dim3 gridDim(1, 4, 16);
  const dim3 threadIdx(thread_id % 8, thread_id / 8 % 2, thread_id / 16);
  const dim3 blockIdx(block_id % 1, block_id / 1 % 4, block_id / 4);
  float *pad_temp_shared = (float *)(shared_buffer + 0);
  float *input1_shared = (float *)(shared_buffer + 1024);
  {
    float *compute = output0;
    {
      float compute1[1];

      compute1[0] = 0.000000e+00f;
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                   (((int)blockIdx.y) * 16)) +
                  (((int)threadIdx.x) * 2))];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                   (((int)threadIdx.y) * 8)) +
                  ((int)threadIdx.x))];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1024)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  1025)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  16)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2048)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  2049)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  32)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3072)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  3073)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  48)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4096)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  4097)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  64)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5120)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  5121)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  80)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6144)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  6145)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  96)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      __syncthreads();
      pad_temp_shared[(((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
                       (((int)threadIdx.x) * 2))] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7168)];
      pad_temp_shared[(
          (((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) +
           (((int)threadIdx.x) * 2)) +
          1)] =
          input0[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 64)) +
                    (((int)blockIdx.y) * 16)) +
                   (((int)threadIdx.x) * 2)) +
                  7169)];
      input1_shared[(((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) +
                     ((int)threadIdx.x))] =
          input1[(((((((int)blockIdx.z) * 1024) + (((int)threadIdx.z) * 128)) +
                    (((int)threadIdx.y) * 8)) +
                   ((int)threadIdx.x)) +
                  112)];
      __syncthreads();
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[((((int)threadIdx.y) * 8) + ((int)threadIdx.x))] *
            input1_shared[(((int)threadIdx.z) * 16)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             16)] *
            input1_shared[((((int)threadIdx.z) * 16) + 1)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             32)] *
            input1_shared[((((int)threadIdx.z) * 16) + 2)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             48)] *
            input1_shared[((((int)threadIdx.z) * 16) + 3)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             64)] *
            input1_shared[((((int)threadIdx.z) * 16) + 4)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             80)] *
            input1_shared[((((int)threadIdx.z) * 16) + 5)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             96)] *
            input1_shared[((((int)threadIdx.z) * 16) + 6)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             112)] *
            input1_shared[((((int)threadIdx.z) * 16) + 7)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             128)] *
            input1_shared[((((int)threadIdx.z) * 16) + 8)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             144)] *
            input1_shared[((((int)threadIdx.z) * 16) + 9)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             160)] *
            input1_shared[((((int)threadIdx.z) * 16) + 10)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             176)] *
            input1_shared[((((int)threadIdx.z) * 16) + 11)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             192)] *
            input1_shared[((((int)threadIdx.z) * 16) + 12)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             208)] *
            input1_shared[((((int)threadIdx.z) * 16) + 13)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             224)] *
            input1_shared[((((int)threadIdx.z) * 16) + 14)]));
      compute1[0] =
          (compute1[0] +
           (pad_temp_shared[(((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) +
                             240)] *
            input1_shared[((((int)threadIdx.z) * 16) + 15)]));
      compute[(((((((int)blockIdx.z) * 512) + (((int)threadIdx.z) * 64)) +
                 (((int)blockIdx.y) * 16)) +
                (((int)threadIdx.y) * 8)) +
               ((int)threadIdx.x))] =
          max((compute1[0] +
               input2[((((int)blockIdx.z) * 8) + ((int)threadIdx.z))]),
              0.000000e+00f);
    }
  }
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_1379_0	type: float	shape: Shape{1, 128, 8,
//8}
//	- name: Constant_2868_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: BatchNormInference_1324_0	type: float	shape: Shape{1,
//128, 8, 8}
// Output:
//	- name: Relu_1395_0	type: float	shape: Shape{1, 128, 8, 8}
//	- name: Add_1387_0	type: float	shape: Shape{1, 128, 8, 8}
// Fused functions:
// Add_float_float_float_cuda_Add_2517<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_1379_0, Constant_2868_0, BatchNormInference_1383_0);
// Add_float_float_float_cuda_Add_1387<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_1383_0, BatchNormInference_1324_0, Add_1387_0);
// Relu_float_float_cuda_Relu_1395<<<dim3(16, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Add_1387_0, Relu_1395_0);
__device__ __forceinline__ static void
FusedKernel_float_float_float_float_float_cuda_Add_Add_Relu_51_block_kernel(
    float *input0, float *input1, float *input2, float *output0, float *output1,
    int thread_id, int block_id, char *shared_buffer) {
  if (thread_id >= 512) {
    return;
  }
  const dim3 blockDim(512, 1, 1);
  const dim3 gridDim(16, 1, 1);
  const dim3 blockIdx(block_id, 0, 0);
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = add(temp0, input2[tid]);
  float temp2 = relu(temp1);
  output1[tid] = temp1;
  output0[tid] = temp2;
}
extern "C" __global__ void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_fused_kernel_126(
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *input11, float *output0, float *output1,
    float *output2, float *output3, float *output4) {
  __shared__ char shared_buffer[1536];

  if ((int)blockIdx.x >= 0 && (int)blockIdx.x <= 63) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121_block_kernel(
        input0, input1, input2, output0, threadIdx.x, blockIdx.x - 0 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 64 && (int)blockIdx.x <= 127) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121_block_kernel(
        input3, input4, input5, output1, threadIdx.x, blockIdx.x - 64 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 128 && (int)blockIdx.x <= 191) {
    Matched_Pattern_float_float_float_float_cuda_Matched_Pattern_3121_block_kernel(
        input6, input7, input8, output2, threadIdx.x, blockIdx.x - 128 + 0,
        shared_buffer);
  } else if ((int)blockIdx.x >= 192 && (int)blockIdx.x <= 207) {
    FusedKernel_float_float_float_float_float_cuda_Add_Add_Relu_51_block_kernel(
        input10, input9, input11, output4, output3, threadIdx.x,
        blockIdx.x - 192 + 0, shared_buffer);
  }
}
extern void
BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_fused_kernel_126_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *input2, float *input3, float *input4,
    float *input5, float *input6, float *input7, float *input8, float *input9,
    float *input10, float *input11, float *output0, float *output1,
    float *output2, float *output3, float *output4) {
  BlockFusionKernel_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_float_cuda_Matched_Pattern_Matched_Pattern_Matched_Pattern_fused_kernel_126<<<
      grids, blocks, mem, stream>>>(
      input0, input1, input2, input3, input4, input5, input6, input7, input8,
      input9, input10, input11, output0, output1, output2, output3, output4);
}
// Node name:	 Elementwise Kernel Fusion
// Input:
//	- name: Convolution_466_0	type: float	shape: Shape{1, 96, 32,
//32}
//	- name: Constant_2978_0	type: float	shape: Shape{1, 96, 32, 32}
// Output:
//	- name: Relu_468_0	type: float	shape: Shape{1, 96, 32, 32}
//	- name: BatchNormInference_467_0	type: float	shape: Shape{1,
//96, 32, 32}
// Fused functions:
// Add_float_float_float_cuda_Add_2013<<<dim3(192, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(Convolution_466_0, Constant_2978_0, BatchNormInference_467_0);
// Relu_float_float_cuda_Relu_468<<<dim3(192, 1, 1), dim3(512, 1, 1), 0,
// 0>>>(BatchNormInference_467_0, Relu_468_0);
extern "C" __launch_bounds__(512) __global__
    void FusedKernel_float_float_float_float_cuda_Add_Relu_0(float *input0,
                                                             float *input1,
                                                             float *output0,
                                                             float *output1) {
  int tid = blockIdx.x * 512 + threadIdx.x;
  float temp0 = add(input0[tid], input1[tid]);
  float temp1 = relu(temp0);
  output1[tid] = temp0;
  output0[tid] = temp1;
}
extern void FusedKernel_float_float_float_float_cuda_Add_Relu_0_Call(
    const dim3 &grids, const dim3 &blocks, unsigned mem, hipStream_t stream,
    float *input0, float *input1, float *output0, float *output1) {
  FusedKernel_float_float_float_float_cuda_Add_Relu_0<<<grids, blocks, mem,
                                                        stream>>>(
      input0, input1, output0, output1);
}
