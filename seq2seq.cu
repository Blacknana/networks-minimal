#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <absl/types/span.h>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <memory>
#include <mma.h>
#include <vector>

template <class T>
static hipError_t LaunchKernel(hipFunction_t f, unsigned grid_x, unsigned block_x,
                             hipStream_t stream, const T &param,
                             unsigned shared = 0) {
  size_t size = sizeof(T);
  void *config[] = {HIP_LAUNCH_PARAM_BUFFER_POINTER, const_cast<T *>(&param),
                    HIP_LAUNCH_PARAM_BUFFER_SIZE, &size, HIP_LAUNCH_PARAM_END};
  return hipModuleLaunchKernel(f, grid_x, 1, 1, block_x, 1, 1, shared, stream, nullptr,
                        config);
}

#define CU_CHECK(error)                                                        \
  {                                                                            \
    if (error != hipSuccess) {                                               \
      const char *error_name;                                                  \
      hipDrvGetErrorName(error, &error_name);                                      \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error, error_name);            \
      exit(1);                                                                 \
    }                                                                          \
  }

#define CUDA_CHECK(error)                                                      \
  {                                                                            \
    if (error != hipSuccess) {                                                \
      fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                   \
      fprintf(stderr, "code: %d, reason: %s\n", error,                         \
              hipGetErrorString(error));                                      \
      exit(1);                                                                 \
    }                                                                          \
  }

namespace seq2seq {

enum Seq2seqScaleParams {
  kEncoderCellNumber = 8,
  kDecoderCellNumber = 4,
  kEncoderTimestep = 100,
  kDecoderTimestep = 30,
  kInputSize = 128,
  kHiddenSize = 128,
  kLstmGateNumber = 4,
};

enum Seq2seqKernelScaleParams {
  kThreadsPerWarp = 32,
  kWarpsPerBlock = 8,
  kColumnsPerBlock = kThreadsPerWarp,
  kGemvBlockNumber = kHiddenSize / kColumnsPerBlock,
  kRowsPerWarp = kHiddenSize / kWarpsPerBlock,
};

#pragma pack(push, 4)
struct CellModel {
  float weights_w[kLstmGateNumber][kInputSize][kHiddenSize];
  float weights_u[kLstmGateNumber][kHiddenSize][kHiddenSize];
  float bias[kLstmGateNumber][kHiddenSize];
};

struct ModelParams {
  CellModel encoder_model[kEncoderCellNumber];
  CellModel decoder_model[kDecoderCellNumber];
};

struct CellState {
  float data[kHiddenSize];
};

struct CellTemp {
  float data[kLstmGateNumber][kHiddenSize];
};

struct CellParams {
  CellState encoder_state_h[kEncoderCellNumber + 1][kEncoderTimestep + 1];
  CellState decoder_state_h[(kDecoderTimestep + 1) * kDecoderCellNumber];
  CellState encoder_state_c[kEncoderCellNumber];
  CellState decoder_state_c[kDecoderCellNumber];
  CellTemp encoder_temp[kEncoderCellNumber];
  CellTemp decoder_temp[kDecoderCellNumber];
};
#pragma pack(pop)
} // namespace seq2seq

using namespace seq2seq;

#pragma pack(push, 1)
struct KernelParams {
  hipDeviceptr_t d_model_params;
  hipDeviceptr_t d_cell_params;
  int step_idx;
  int cell_idx;
};

struct KernelParams0 {
  hipDeviceptr_t d_model_params;
  hipDeviceptr_t d_cell_params;
};
#pragma pack(pop)

class Seq2seqWave {
public:
  Seq2seqWave();
  void InitCellParams(hipDeviceptr_t d_cell_params);
  void EncoderCompute(int wave_size, KernelParams kernel_params);
  void DecoderCompute(KernelParams kernel_params);
  void Finalize();

private:
  hipDevice_t cu_device_;
  hipCtx_t cu_context_;
  hipFunction_t cu_wave_encoder_;
  hipFunction_t cu_decoder_;
  hipFunction_t cu_decoder_layer0_step0_;
};

class Seq2seq {
public:
  explicit Seq2seq(absl::Span<const float> src_model);
  bool Initialize(absl::Span<const float> input);
  void Solve();
  bool Fetch(absl::Span<float> output);
  void Finalize();

private:
  Seq2seqWave wave_;
  hipDeviceptr_t d_model_params_;
  hipDeviceptr_t d_cell_params_;
  hipDeviceptr_t d_input_;
  hipDeviceptr_t d_output_;
};

__global__ void seq2seq_wave_encoder(ModelParams *d_model_params,
                                     CellParams *d_cell_params,
                                     int step_start_num, int layer_start_num);
__global__ void seq2seq_decoder(ModelParams *d_model_params,
                                CellParams *d_cell_params, int cell_idx,
                                int step_idx);
__global__ void seq2seq_decoder_layer0_step0(ModelParams *d_model_params,
                                             CellParams *d_cell_params);

int main() {
  std::vector<float> input(sizeof(CellState) * kEncoderTimestep /
                           sizeof(float));
  std::vector<float> model(sizeof(ModelParams));
  std::vector<float> output_buffer(sizeof(CellState) * kEncoderTimestep /
                                   sizeof(float));
  absl::Span<float> output(output_buffer);
  auto network = new Seq2seq(model);

  enum { kWarmUp = 200, kLoop = 1000 };
  // Warm-up, 100 times
  for (int i = 0; i < kWarmUp; i++) {
    network->Initialize(input);
    network->Solve();
    network->Fetch(output);
  }
  double min_ms = std::numeric_limits<double>::max();
  double max_ms = std::numeric_limits<double>::min();
  double total_ms = 0.00000f;
  for (int i = 0; i < kLoop; i++) {
    auto start = std::chrono::steady_clock::now();
    network->Initialize(input);
    network->Solve();
    network->Fetch(output);
    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double, std::micro> elapsed = end - start;
    double iteration_ms = elapsed.count();
    printf("Iteration time %f us\n", iteration_ms);
    min_ms = std::min(iteration_ms, min_ms);
    max_ms = std::max(iteration_ms, max_ms);
    total_ms = total_ms + iteration_ms;
  }
  printf("Sumamry: [min, max, mean] = [%f, %f, %f] us\n", min_ms, max_ms,
         total_ms / kLoop);

  network->Finalize();

  return 0;
}

Seq2seqWave::Seq2seqWave() {
  CU_CHECK(hipInit(0));
  CU_CHECK(hipDeviceGet(&cu_device_, 0));
  CU_CHECK(hipCtxCreate(&cu_context_, 0, cu_device_));
  CUDA_CHECK(hipGetFuncBySymbol(&cu_wave_encoder_,
                                 (const void *)seq2seq_wave_encoder));
  CUDA_CHECK(hipGetFuncBySymbol(&cu_decoder_, (const void *)seq2seq_decoder));
  CUDA_CHECK(hipGetFuncBySymbol(&cu_decoder_layer0_step0_,
                                 (const void *)seq2seq_decoder_layer0_step0));
}

void Seq2seqWave::Finalize() { CU_CHECK(hipCtxDestroy(cu_context_)); }

void Seq2seqWave::EncoderCompute(int wave_size, KernelParams kernel_params) {
  CU_CHECK(LaunchKernel(cu_wave_encoder_, kGemvBlockNumber * wave_size,
                        kHiddenSize, 0, kernel_params));
}

void Seq2seqWave::DecoderCompute(KernelParams kernel_params) {
  if (kernel_params.step_idx == 0 && kernel_params.cell_idx == 0) {
    KernelParams0 kernel_params0 = {kernel_params.d_model_params,
                                    kernel_params.d_cell_params};
    CU_CHECK(LaunchKernel(cu_decoder_layer0_step0_, kGemvBlockNumber,
                          kHiddenSize, 0, kernel_params0));
  } else {
    CU_CHECK(LaunchKernel(cu_decoder_, kGemvBlockNumber, kHiddenSize, 0,
                          kernel_params));
  }
}

void Seq2seqWave::InitCellParams(hipDeviceptr_t d_cell_params) {
  CU_CHECK(hipMemsetD32(d_cell_params, 0.000000e+00f,
                       sizeof(CellParams) / sizeof(float)));
}

Seq2seq::Seq2seq(absl::Span<const float> src_model) {
  CU_CHECK(hipMalloc(&d_model_params_, sizeof(ModelParams)));
  CU_CHECK(hipMalloc(&d_cell_params_, sizeof(CellParams)));
  CU_CHECK(
      hipMemcpyHtoD(d_model_params_, src_model.data(), sizeof(ModelParams)));

  d_input_ = d_cell_params_ + sizeof(CellState);
  d_output_ = d_cell_params_ + sizeof(CellParams::encoder_state_h) +
              sizeof(CellParams::decoder_state_h) -
              kDecoderTimestep * sizeof(CellState);
}

bool Seq2seq::Initialize(absl::Span<const float> input) {
  if (input.size() != sizeof(CellState) * kEncoderTimestep / sizeof(float)) {
    return false;
  }

  wave_.InitCellParams(d_cell_params_);
  CU_CHECK(hipMemcpyHtoD(d_input_, input.data(),
                        sizeof(CellState) * kEncoderTimestep));
  return true;
}

void Seq2seq::Solve() {
  const int max_wave_size = std::min(kEncoderCellNumber, kEncoderTimestep);
  const int max_wave_number = kEncoderCellNumber + kEncoderTimestep - 1;
  for (int wave_idx = 1; wave_idx <= max_wave_number; ++wave_idx) {
    int wave_size =
        (wave_idx < std::max(kEncoderCellNumber, kEncoderTimestep))
            ? std::min(wave_idx, max_wave_size)
            : (max_wave_size -
               (wave_idx - std::max(kEncoderCellNumber, kEncoderTimestep)));
    int step_start_num =
        (wave_idx < kEncoderTimestep) ? wave_idx : kEncoderTimestep;
    int layer_start_num =
        (wave_idx < kEncoderTimestep) ? 1 : (wave_idx - kEncoderTimestep + 1);

    KernelParams kernel_params = {d_model_params_, d_cell_params_,
                                  step_start_num, layer_start_num};
    wave_.EncoderCompute(wave_size, kernel_params);
  }

  for (int step_idx = 1; step_idx <= kDecoderTimestep; ++step_idx) {
    for (int cell_idx = 1; cell_idx <= kDecoderCellNumber; ++cell_idx) {
      KernelParams kernel_params = {d_model_params_, d_cell_params_, step_idx,
                                    cell_idx};
      wave_.DecoderCompute(kernel_params);
    }
  }
}

bool Seq2seq::Fetch(absl::Span<float> output) {
  if (output.size() != sizeof(CellState) * kDecoderTimestep / sizeof(float)) {
    return false;
  }

  CU_CHECK(hipMemcpyDtoH(output.data(), d_output_,
                        sizeof(CellState) * kDecoderTimestep));
  return true;
}

void Seq2seq::Finalize() {
  CU_CHECK(hipFree(d_model_params_));
  CU_CHECK(hipFree(d_cell_params_));
  wave_.Finalize();
}

__device__ static inline float sigmoid(float x) {
  return 1.000000e+00f / (1.000000e+00f + __expf(0.000000e+00f - x));
}

__device__ static inline void
seq2seq_compute(CellState *d_input, CellState *d_input_state_h,
                CellState *d_output_state_h, CellState *d_state_c,
                CellTemp *d_temp, CellModel *d_model) {
  const int warp_idx = threadIdx.x / kThreadsPerWarp;
  const int lane_idx = threadIdx.x % kThreadsPerWarp;
  const int col_idx =
      (blockIdx.x % kGemvBlockNumber) * kColumnsPerBlock + lane_idx;

  if (warp_idx == 0) {
    for (int i = 0; i < kLstmGateNumber; ++i) {
      d_temp->data[i][col_idx] = 0.000000e+00f;
    }
  }
  __syncthreads();

  float temp[kLstmGateNumber] = {0.000000e+00f, 0.000000e+00f, 0.000000e+00f,
                                 0.000000e+00f};
  const int row_start_idx = kRowsPerWarp * warp_idx;
  const int row_end_idx = row_start_idx + kRowsPerWarp;
  for (int row_idx = row_start_idx; row_idx < row_end_idx; ++row_idx) {
    float input_data = d_input->data[row_idx];
    float state_h_data = d_input_state_h->data[row_idx];
    for (int i = 0; i < kLstmGateNumber; ++i) {
      temp[i] =
          fma(d_model->weights_w[i][row_idx][col_idx], input_data, temp[i]);
    }
    for (int i = 0; i < kLstmGateNumber; ++i) {
      temp[i] =
          fma(d_model->weights_u[i][row_idx][col_idx], state_h_data, temp[i]);
    }
  }

  for (int i = 0; i < kLstmGateNumber; ++i) {
    atomicAdd(&d_temp->data[i][col_idx], temp[i]);
  }
  __syncthreads();

  if (warp_idx == 0) {
    float input_gate_x = d_temp->data[0][col_idx] + d_model->bias[0][col_idx];
    float input_gate_y = d_temp->data[1][col_idx] + d_model->bias[1][col_idx];
    float forget_gate = d_temp->data[2][col_idx] + d_model->bias[2][col_idx];
    float output_gate = d_temp->data[3][col_idx] + d_model->bias[3][col_idx];
    input_gate_x = sigmoid(input_gate_x);
    input_gate_y = tanh(input_gate_y);
    output_gate = sigmoid(output_gate);
    forget_gate =
        sigmoid(forget_gate + 1.000000e+00f) * d_state_c->data[col_idx];
    d_state_c->data[col_idx] = fma(input_gate_x, input_gate_y, forget_gate);
    d_output_state_h->data[col_idx] =
        (tanh(d_state_c->data[col_idx])) * output_gate;
  }
}

__global__ void __launch_bounds__(128, 4)
    seq2seq_wave_encoder(ModelParams *d_model_params, CellParams *d_cell_params,
                         int step_start_num, int layer_start_num) {
  const int cell_idx = layer_start_num + blockIdx.x / kGemvBlockNumber;
  const int step_idx = step_start_num - blockIdx.x / kGemvBlockNumber;
  CellState *d_input = &d_cell_params->encoder_state_h[cell_idx - 1][step_idx];
  CellState *d_input_state_h =
      &d_cell_params->encoder_state_h[cell_idx][step_idx - 1];
  CellState *d_output_state_h =
      &d_cell_params->encoder_state_h[cell_idx][step_idx];
  CellState *d_state_c = &d_cell_params->encoder_state_c[cell_idx - 1];
  CellTemp *d_temp = &d_cell_params->encoder_temp[cell_idx - 1];
  CellModel *d_model = &d_model_params->encoder_model[cell_idx - 1];
  seq2seq_compute(d_input, d_input_state_h, d_output_state_h, d_state_c, d_temp,
                  d_model);
}

__global__ void __launch_bounds__(128, 4)
    seq2seq_decoder(ModelParams *d_model_params, CellParams *d_cell_params,
                    int step_idx, int cell_idx) {
  CellState *d_input =
      &d_cell_params->decoder_state_h[step_idx * kDecoderCellNumber +
                                      (cell_idx - 1) - 1];
  CellState *d_input_state_h =
      &d_cell_params->decoder_state_h[(step_idx - 1) * kDecoderCellNumber +
                                      cell_idx - 1];
  CellState *d_output_state_h =
      &d_cell_params
           ->decoder_state_h[step_idx * kDecoderCellNumber + cell_idx - 1];
  CellState *d_state_c = &d_cell_params->decoder_state_c[cell_idx - 1];
  CellTemp *d_temp = &d_cell_params->decoder_temp[cell_idx - 1];
  CellModel *d_model = &d_model_params->decoder_model[cell_idx - 1];
  seq2seq_compute(d_input, d_input_state_h, d_output_state_h, d_state_c, d_temp,
                  d_model);
}

__global__ void __launch_bounds__(128, 4)
    seq2seq_decoder_layer0_step0(ModelParams *d_model_params,
                                 CellParams *d_cell_params) {
  CellState *d_input = &d_cell_params->encoder_state_c[kEncoderCellNumber - 1];
  CellState *d_input_state_h = &d_cell_params->decoder_state_h[0];
  CellState *d_output_state_h =
      &d_cell_params->decoder_state_h[kDecoderCellNumber];
  CellState *d_state_c = &d_cell_params->decoder_state_c[0];
  CellTemp *d_temp = &d_cell_params->decoder_temp[0];
  CellModel *d_model = &d_model_params->decoder_model[0];
  seq2seq_compute(d_input, d_input_state_h, d_output_state_h, d_state_c, d_temp,
                  d_model);
}